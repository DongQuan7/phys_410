#include "hip/hip_runtime.h"
/*
 * add-passb.cu
 */
/* cf. Jason Sanders, Edward Kandrot. CUDA by Example: An Introduction to General-Purpose GPU Programming */
/* 3.2.2 A Kernel Call */
#include <stdio.h>
#include "common/errors.h"

__global__ void add(int a, int b, int *c) {
	*c = a + b;
}

int main(void) {
	int c;
	int *dev_c;

	HANDLE_ERROR(
		     hipMalloc((void**)&dev_c, sizeof(int))
		     );

	add<<<1,1>>>(2,7,dev_c);

	HANDLE_ERROR(
		     hipMemcpy(&c, dev_c, sizeof(int),hipMemcpyDeviceToHost)
		     );

	printf("2 + 7 =%d\n",c);
	hipFree(dev_c);

	return 0;
}
