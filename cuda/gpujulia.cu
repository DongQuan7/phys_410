#include "hip/hip_runtime.h"
/* cf. Jason Sanders, Edward Kandrot. CUDA by Example: An Introduction to General-Purpose GPU Programming */
/* 
** Chapter 4 Parallel Programming in CUDA C
** 4.2 CUDA Parallel Programming
** 4.2.1 Summing Vectors
*/
#include <stdio.h>
#include "common/errors.h"
#include "common/cpu_bitmap.h"

#define DIM 1000

struct hipComplex {
  float r;
  float i;
  __device__ hipComplex( float a, float b) : r(a), i(b) {}
  __device__ float magnitude2( void ) {
    return r * r + i * i;
  }
  __device__ hipComplex operator*(const hipComplex& a) {
    return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
  }
  __device__ hipComplex operator+(const hipComplex& a) {
    return hipComplex(r+a.r, i+a.i);
  }
};

__device__ int julia( int x, int y) {
  const float scale = 1.5;
  float jx = scale * (float)(DIM/2 - x)/(DIM/2);
  float jy = scale * (float)(DIM/2 - y)/(DIM/2);

  hipComplex c(-0.8,0.154);
  hipComplex a(jx,jy);

  int i = 0;
  for (i=0; i<200; i++) {
    a = a*a + c;
    if (a.magnitude2() > 1000)
      return 0; // return 0 if it is not in set
  }
  return 1; // return 1 if point is in set
}

__global__ void kernel( unsigned char *ptr) {
  // map from threadIdx/BlockIdx to pixel position
  int x = blockIdx.x;
  int y = blockIdx.y;
  int offset = x+y*gridDim.x;

  // now calculate the value at that position
  int juliaValue = julia(x,y);
  ptr[offset*4 + 0] = 255 * juliaValue;  // red if julia() returns 1, black if pt. not in set
  ptr[offset*4 + 1] = 0;
  ptr[offset*4 + 2] = 0;
  ptr[offset*4 + 3] = 255;
}
  

int main(void) {
  CPUBitmap bitmap( DIM, DIM );
  unsigned char *dev_bitmap;

  HANDLE_ERROR( hipMalloc( (void**)&dev_bitmap, bitmap.image_size() ) );

  dim3 grid(DIM,DIM);

  kernel<<<grid,1>>>(dev_bitmap);

  HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(),
			  dev_bitmap,
			  bitmap.image_size(),
			  hipMemcpyDeviceToHost ));
  bitmap.display_and_exit();

  HANDLE_ERROR( hipFree(dev_bitmap));
}
