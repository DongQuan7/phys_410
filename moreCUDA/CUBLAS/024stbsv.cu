/**
 * @file   : 024stbsv.cu
 * @brief  : cublasStbsv - solve the triangular banded linear system  
 * uses CUDA Unified Memory (Management)
 * @author : Ernest Yeung	ernestyalumni@gmail.com
 * @date   : 20170417
 * @ref    :  cf. https://developer.nvidia.com/sites/default/files/akamai/cuda/files/Misc/mygpu.pdf
 * 
 * If you find this code useful, feel free to donate directly and easily at this direct PayPal link: 
 * 
 * https://www.paypal.com/cgi-bin/webscr?cmd=_donations&business=ernestsaveschristmas%2bpaypal%40gmail%2ecom&lc=US&item_name=ernestyalumni&currency_code=USD&bn=PP%2dDonationsBF%3abtn_donateCC_LG%2egif%3aNonHosted 
 * 
 * which won't go through a 3rd. party such as indiegogo, kickstarter, patreon.  
 * Otherwise, I receive emails and messages on how all my (free) material on 
 * physics, math, and engineering have helped students with their studies, 
 * and I know what it's like to not have money as a student, but love physics 
 * (or math, sciences, etc.), so I am committed to keeping all my material 
 * open-source and free, whether or not 
 * sufficiently crowdfunded, under the open-source MIT license: 
 * 	feel free to copy, edit, paste, make your own versions, share, use as you wish.  
 *  Just don't be an asshole and not give credit where credit is due.  
 * Peace out, never give up! -EY
 * 
 * */
// COMPILATION TIP:
// nvcc -std=c++11 -arch='sm_52' 024stbsv.cu -lcublas -o 024stbsv.exe
#include <iostream>
#include <hip/hip_runtime.h>
#include "hipblas.h"

constexpr const int n =6;			// number of rows and columns of a 
constexpr const int k =1;				// number of subdiagonals

//__device__ __managed__ float *A;  	// nxn matrix A on CUDA Unified (managed) memory Segmentation fault (core dumped)
__device__ __managed__ float A[n*n];  	// nxn matrix A on CUDA Unified (managed) memory 

//__device__ __managed__ float *x;	// n-vector x on CUDA Unified (managed) memory Segmentation fault (core dumped)
__device__ __managed__ float b[n];	// n-vector x on CUDA Unified (managed) memory 

__device__ __managed__ float d_A[n*n];
__device__ __managed__ float d_b[n];


int main(void) {
	hipError_t cudaStat;					// cudaMalloc status
	hipblasStatus_t stat;					// CUBLAS functions status
	hipblasHandle_t handle;					// CUBLAS context
	int i,j;							// lower triangle of a:
	
	// main diagonal and subdiagonals of A in rows:
	int ind=11;
	int d_ind=11;
	// main diagonal: 11, 12,13,14,15,16 in row 0
	for (i=0; i<n; i++) {
		A[i*n]=(float)ind++;  
		d_A[i*n]=(float)d_ind++; 
	}
	
	// first subdiagonal: 17, 18, 19, 20, 21 in row 1
	for (i=0;i<n-1;i++) {
		A[i*n+1]=(float)ind++; 
		d_A[i*n+1]=(float)d_ind++;
	}
		
	for (i=0; i<n;i++) {
		b[i]=1.0f; 
		d_b[i]=1.0f;
	}					// b={1,1,1,1,1,1}^T

	for (j=0;j<n;j++){
		for (i=0;i<n;i++){
			std::cout << A[i + n*j] << " "; 
	} 
		std::cout << std::endl; }

	for (i=0;i<n; i++) {
		std::cout << b[i] << " "; } std::cout << std::endl;

		
	stat = hipblasCreate(&handle);		// initialize CUBLAS context
//	stat = cublasSetMatrix(n,n,sizeof(*A),A,n,d_A,n);  	// this works
//	stat = cublasSetVector(n,sizeof(*b),b,1,d_b,1);		// this works

	for (j=0;j<n;j++){
		for (i=0;i<n;i++){
			std::cout << d_A[i + n*j] << " "; 
	} 
		std::cout << std::endl; }


	// solve a triangular banded linear system: Ax=b;
	// the solution x overwrite the right hand side (RHS) b;
	// A - nxn banded lower triangular matrix; b - n-vector

//	stat=cublasStbsv(handle,CUBLAS_FILL_MODE_LOWER,CUBLAS_OP_N,
//							CUBLAS_DIAG_NON_UNIT, n,k, d_A,n,d_b,1);
	stat=hipblasStbsv(handle,HIPBLAS_FILL_MODE_LOWER,HIPBLAS_OP_N,
							HIPBLAS_DIAG_NON_UNIT, n,k, d_A,n, d_b,1);

	stat = hipblasGetVector(n,sizeof(float),d_b,1,b,1);

	// print the solution
	std::cout << "solution : " << std::endl; 	// print x after Stbsv
	for(j=0;j<n;j++) 
	{
		std::cout << b[j] << std::endl; }
	hipblasDestroy(handle);	// destroy CUBLAS context	
	
//	cudaDeviceReset();
	return EXIT_SUCCESS;

}
