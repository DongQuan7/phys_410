/**
 * @file   : SVD_CC.cu
 * @brief  : Simple example in C of singular value decomposition, but with singular vectors
 * 				compute A = U*S*VT
 * @author : Ernest Yeung	ernestyalumni@gmail.com
 * @date   : 20170703
 * @ref    :  cf. http://docs.nvidia.com/cuda/cusolver/index.html#svd_examples
 * 
 * If you find this code useful, feel free to donate directly and easily at this direct PayPal link: 
 * 
 * https://www.paypal.com/cgi-bin/webscr?cmd=_donations&business=ernestsaveschristmas%2bpaypal%40gmail%2ecom&lc=US&item_name=ernestyalumni&currency_code=USD&bn=PP%2dDonationsBF%3abtn_donateCC_LG%2egif%3aNonHosted 
 * 
 * which won't go through a 3rd. party such as indiegogo, kickstarter, patreon.  
 * Otherwise, I receive emails and messages on how all my (free) material on 
 * physics, math, and engineering have helped students with their studies, 
 * and I know what it's like to not have money as a student, but love physics 
 * (or math, sciences, etc.), so I am committed to keeping all my material 
 * open-source and free, whether or not 
 * sufficiently crowdfunded, under the open-source MIT license: 
 * 	feel free to copy, edit, paste, make your own versions, share, use as you wish.  
 *  Just don't be an asshole and not give credit where credit is due.  
 * Peace out, never give up! -EY
 * 
 * */
#include <iostream> 	// std::cout
#include <iomanip> 		// std::setprecision 

#include <assert.h>
#include <hip/hip_runtime.h>  // cudaError_t
#include <hipsolver.h> // Dn = dense (matrices)

#include <hip/hip_complex.h> // cuComplex, cuDoubleComplex

void printMatrix(int m, int n, const hipDoubleComplex *A, int lda, const char* name) 
{
	std::cout << name << std::endl;
	for (int row =0; row <m; row++) {
		for (int col =0 ; col <n ; col++) {
			hipDoubleComplex Areg = A[row + col*lda]; 
			std::cout << std::setprecision(9) << Areg.x << "+i" << Areg.y << " " ; 
		}
		std::cout << std::endl;
	}
}

template <typename TT>
void print1darr(const int N, const TT *A, int lda, const char* name) 
{
	std::cout << name << std::endl;
	for (int row =0; row < N; row++) {
		TT Areg = A[row *lda]; 
		std::cout << Areg << " "  ; 
	}
	std::cout << std::endl;
}

void create_linearval_C_matrix(const int M, const int N, hipDoubleComplex *A) {
	double ind_CC = 0.1; // value to scale the imaginary parts values by
	for (int i=0; i < M; i++) { 
		for (int j=0; j<N; j++) { 
			A[i+j*M].x = ((double) (i+1+ M*j));
			A[i+j*M].y = ind_CC * ( (double) i+1 + M*j); 
		}
	}
	
}

int main(int argc, char* argv[]) {
	hipsolverHandle_t cusolverH = NULL;
	hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;

	hipError_t cudaStat = hipSuccess;  // cudaSuccess=0, cf. http://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html#axzz4lEpqZl2L
	
	constexpr const int M = 4; // number of rows 
	constexpr const int N = 2; // number of columns  
	constexpr const int lda = M; 

	hipDoubleComplex A[M*N]; 
	create_linearval_C_matrix(M,N,A); 
	printMatrix(M,N,A,M,"A"); 
	
	hipDoubleComplex U[M*M]; // M-by-M unitary matrix
	hipDoubleComplex VT[N*N]; // N-by-N unitary matrix
	double S[N]; // singular value
	
	hipDoubleComplex *d_A = nullptr; 
	double *d_S = nullptr; 
	hipDoubleComplex *d_U = nullptr; 
	hipDoubleComplex *d_VT = nullptr; 
	int *devInfo = nullptr; 
	hipDoubleComplex *d_work = nullptr; 
	double *d_rwork = nullptr; 
	
	int lwork = 0;
	
	
// step 1: create cusolverDn/cublas handle 
	cusolver_status = hipsolverDnCreate(&cusolverH);
	assert (HIPSOLVER_STATUS_SUCCESS == cusolver_status);

// step 2: copy A and B to device
	cudaStat = hipMalloc((void**)&d_A , sizeof(hipDoubleComplex)*M*N);
	assert(hipSuccess == cudaStat);
	cudaStat = hipMalloc((void**)&d_S , sizeof(double)*N);
	assert(hipSuccess == cudaStat);
	cudaStat = hipMalloc((void**)&d_U , sizeof(hipDoubleComplex)*M*M);
	assert(hipSuccess == cudaStat);
	cudaStat = hipMalloc((void**)&d_VT , sizeof(hipDoubleComplex)*N*N);
	assert(hipSuccess == cudaStat);
	cudaStat = hipMalloc((void**)&devInfo, sizeof(int));
	assert(hipSuccess == cudaStat);
	
		
	cudaStat = hipMemcpy(d_A, A, sizeof(hipDoubleComplex)*M*N, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);

// step 3: query working space of SVD 
	cusolver_status = hipsolverDnZgesvd_bufferSize(
		cusolverH,
		M,
		N,
		&lwork );
	assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);
	
	cudaStat = hipMalloc((void**)&d_work , sizeof(hipDoubleComplex)*lwork);
	assert(hipSuccess == cudaStat);

// step 4: compute SVD 
	signed char jobu = 'A'; // all m columns of U
	signed char jobvt = 'A'; // all n columns of VT
	cusolver_status = hipsolverDnZgesvd(
		cusolverH,
		jobu,
		jobvt,
		M,
		N,
		d_A,
		lda,
		d_S,
		d_U,
		M, 	// ldu
		d_VT,
		N, 	// ldvt,
		d_work,
		lwork, 
		d_rwork,
		devInfo);
	cudaStat = hipDeviceSynchronize();
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
	assert(hipSuccess == cudaStat);
	cudaStat = hipMemcpy(U,d_U, sizeof(hipDoubleComplex)*lda*M,hipMemcpyDeviceToHost); 
	assert(hipSuccess == cudaStat);
	cudaStat = hipMemcpy(VT,d_VT, sizeof(hipDoubleComplex)*N*N,hipMemcpyDeviceToHost); 
	assert(hipSuccess == cudaStat);
	cudaStat = hipMemcpy(S,d_S, sizeof(double)*N,hipMemcpyDeviceToHost); 
	assert(hipSuccess == cudaStat);
	
	
	std::cout << " S = (matlab base-1) " << std::endl; 
	print1darr(N, S, 1, "S");
	std::cout << "=====" << std::endl; 
	
	printf("U = (matlab base-1)\n");
	printMatrix(M, M, U, M, "U");
	printf("=====\n");
	
	printf("VT = (matlab base-1)\n");
	printMatrix(N, N, VT, N, "VT");
	printf("=====\n");
	
	
	


	
// free resources
	if (d_A		) hipFree(d_A);
	if (d_S		) hipFree(d_S);
	if (d_U 	) hipFree(d_U);
	if (d_VT	) hipFree(d_VT);
	if (devInfo ) hipFree(devInfo);
	if (d_work	) hipFree(d_work);
	if (d_rwork	) hipFree(d_rwork);
	
	if (cusolverH) hipsolverDnDestroy(cusolverH);
	
	
	

	hipDeviceReset();
	return 0;




}
