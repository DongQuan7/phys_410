#include "hip/hip_runtime.h"
/**
 * @file   : SVD_singular_values_only_unified.cu
 * @brief  : Example of comparing singular value decomposition of a real matrix  
 * I compare this implementation, that uses CUDA Unified Memory Management, against that by OrangeOwl
 *
 * 				compute A = U*S*VT
 * @author : Ernest Yeung	ernestyalumni@gmail.com
 * @date   : 20170628
 * @ref    :  cf. https://github.com/OrangeOwlSolutions/Linear-Algebra/wiki/SVD-of-a-real-matrix
 * 
 * If you find this code useful, feel free to donate directly and easily at this direct PayPal link: 
 * 
 * https://www.paypal.com/cgi-bin/webscr?cmd=_donations&business=ernestsaveschristmas%2bpaypal%40gmail%2ecom&lc=US&item_name=ernestyalumni&currency_code=USD&bn=PP%2dDonationsBF%3abtn_donateCC_LG%2egif%3aNonHosted 
 * 
 * which won't go through a 3rd. party such as indiegogo, kickstarter, patreon.  
 * Otherwise, I receive emails and messages on how all my (free) material on 
 * physics, math, and engineering have helped students with their studies, 
 * and I know what it's like to not have money as a student, but love physics 
 * (or math, sciences, etc.), so I am committed to keeping all my material 
 * open-source and free, whether or not 
 * sufficiently crowdfunded, under the open-source MIT license: 
 * 	feel free to copy, edit, paste, make your own versions, share, use as you wish.  
 *  Just don't be an asshole and not give credit where credit is due.  
 * Peace out, never give up! -EY
 * 
 * */

/*
 * How to compile (assume cuda is installed at /usr/local/cuda/)
 * 	 nvcc -c -I/usr/local/cuda/include svd_example.cpp
 * 	 g++ -fopenmp -o a.out svd_example.o -L/usr/local/cuda/lib64 -lcudart -lcublas -lcusolver
 * 
 * EY : 20170628 This also worked for me
 * nvcc -std=c++11 -arch='sm_52' -lcudart -lcublas -lcusolver SVD_vectors_unified.cu -o SVD_unified.exe
 * */
 
#include <iostream> 	// std::cout
#include <iomanip> 		// std::setprecision 

#include <math.h> 	// sqrt

#include <assert.h> // assert
#include <hip/hip_runtime.h>  // hipError_t
#include <hipblas.h>
#include <hipsolver.h> // Dn = dense (matrices)

#include "gputimer.h" // GpuTimer

constexpr const int m = 1000;
constexpr const int n = 1000;
constexpr const int lda = m;

__device__ __managed__ double A[m*n] ;
__device__ __managed__ double U[lda*m]; // m-by-m unitary matrix
__device__ __managed__ double VT[n*n]; // n-by-n unitary matrix
__device__ __managed__ double S[n]; 	// singular value
__device__ __managed__ int *devInfo = nullptr; 
__device__ __managed__ double *d_rwork = NULL; 


// Looks like this is for boilerplate and it looks like that 
/* 
 * lda = stride
 * it's in "column-major" order; cuSOLVER assumes for dense matrices COLUMN-major order
 * cf. http://docs.nvidia.com/cuda/cusolver/index.html#format-dense-matrix
 * */
void printMatrix(int m, int n, const double *A, int lda, const char* name) 
{
	std::cout << name << std::endl;
	for (int row =0; row <m; row++) {
		for (int col =0 ; col <n ; col++) {
			float Areg = A[row + col*lda]; 
			std::cout << Areg << " " ; 
		}
		std::cout << std::endl;
	}
}

int main(int argc, char* argv[]) {

	hipsolverHandle_t cusolverH = NULL;
	hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;  
	
	// --- cuSOLVER input/output parameters/arrays
	// working space, <type> array of size lwork
	double *d_work = NULL;
	// size of working array work
	int lwork = 0;
	
// step 1: create cusolverDn/cublas handle, CUDA solver initialization
	cusolver_status = hipsolverDnCreate(&cusolverH);
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

	// --- Setting the boilerplate, initialization values
	for (int i=0; i<m; i++) {
		for (int j=0; j < n; j++) {
			A[i+j*m] = (i*i + j) * sqrt((double)(i+j));
		}
	}

// step 2: query working space of SVD	
	cusolver_status = hipsolverDnSgesvd_bufferSize( cusolverH, m, n, &lwork);
	assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);
	std::cout << " \n lwork = " << lwork << std::endl << std::endl;
	hipMalloc((void**)&d_work, sizeof(double)*lwork);

// step 4: compute SVD
	GpuTimer timer;

	// --- CUDA SVD execution - Singular values only
		/**
	 * jobu, input
	 * @brief specifies options for computing all or part of the matrix U:= 'A': all m columns of U are returned in array 
	 * 	U:='S': the first min(m,n) columns of U (the left singular vectors) are returned in the array
	 * 	U:='O': the first min(m,n) columns of U (the left singular vectors) are overwritten on the array A; 
	 * 	 = 'N': no columns (no left singular vectors) are computed.  
	 * 
	 * */
	timer.Start();
	cusolver_status = hipsolverDnDgesvd(cusolverH,'N','N', m,n, A, lda, 
		S,U, 
		lda, // ldu
		VT, 
		n, // ldvt
		d_work,lwork, d_rwork, devInfo);
	timer.Stop();
	std::cout << " Calculation of singular values only : " << std::setprecision(7) << timer.Elapsed() << " ms " << std::endl;
	
	
	// --- CUDA SVD execution - Full SVD	
	
	timer.Start();

	cusolver_status = hipsolverDnDgesvd(cusolverH,'A','A', m,n, A, lda, 
		S,U, 
		lda, // ldu
		VT, 
		n, // ldvt
		d_work,lwork, d_rwork, devInfo);
	timer.Stop();
	
	std::cout << " Calculation of the full SVD calculation : " << std::setprecision(7) << timer.Elapsed() << " ms " << std::endl;
	
	
// free resources
	if (cusolverH) hipsolverDnDestroy(cusolverH);
	if (d_work) hipFree(d_work);
	
	hipDeviceReset();
	return 0;

}
