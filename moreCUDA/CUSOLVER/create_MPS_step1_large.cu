#include "hip/hip_runtime.h"
/**
 * @file   : create_MPS_step1_large.cu
 * @brief  : 2 Steps of creating Matrix Product State (MPS) consisting of 2 routines; singular value decomposition (SVD) and matrix multiplication (CUBLAS)
 * 				This is an example for large L (L=sites), so 2^L states possible
 * @author : Ernest Yeung	ernestyalumni@gmail.com
 * @date   : 20170704
 * @ref    :  cf. http://docs.nvidia.com/cuda/cusolver/index.html#svd_examples
 * 
 * If you find this code useful, feel free to donate directly and easily at this direct PayPal link: 
 * 
 * https://www.paypal.com/cgi-bin/webscr?cmd=_donations&business=ernestsaveschristmas%2bpaypal%40gmail%2ecom&lc=US&item_name=ernestyalumni&currency_code=USD&bn=PP%2dDonationsBF%3abtn_donateCC_LG%2egif%3aNonHosted 
 * 
 * which won't go through a 3rd. party such as indiegogo, kickstarter, patreon.  
 * Otherwise, I receive emails and messages on how all my (free) material on 
 * physics, math, and engineering have helped students with their studies, 
 * and I know what it's like to not have money as a student, but love physics 
 * (or math, sciences, etc.), so I am committed to keeping all my material 
 * open-source and free, whether or not 
 * sufficiently crowdfunded, under the open-source MIT license: 
 * 	feel free to copy, edit, paste, make your own versions, share, use as you wish.  
 *  Just don't be an asshole and not give credit where credit is due.  
 * Peace out, never give up! -EY
 * 
 * */

#include <iostream> 	// std::cout
#include <iomanip> 		// std::setprecision 

#include <array> // std::array

#include <hip/hip_runtime.h>  // hipError_t
#include <hipsolver.h> // Dn = dense (matrices)

#include <hip/hip_complex.h> // hipComplex, hipDoubleComplex

#include "gputimer.h" // GpuTimer



/* ****************************************************************** */
/* ****** "BOILERPLATE" routines for creating arbitrary initialization 
 * values, print out for human reaidng of results *********************
/* ****************************************************************** */

void printMatrix(int m, int n, const hipDoubleComplex *A, int lda, const char* name) 
{
	std::cout << name << std::endl;
	for (int row =0; row <m; row++) {
		for (int col =0 ; col <n ; col++) {
			hipDoubleComplex Areg = A[row + col*lda]; 
			std::cout << std::setprecision(5) << Areg.x << "+i" << Areg.y << " " ; 
		}
		std::cout << std::endl;
	}
}

template <typename TT>
void print1darr(const int N, const TT *A, int lda, const char* name) 
{
	std::cout << name << std::endl;
	for (int row =0; row < N; row++) {
		TT Areg = A[row *lda]; 
		std::cout << Areg.x << "+i"<< Areg.y << " " ; 
//		std::cout << Areg << " ";
	}
	std::cout << std::endl;
}

/* ****************************************************************** */
/* ****** END of boilerplate ******************************************/
/* ****************************************************************** */

/* ****************************************************************** */
/* ****** MAIN routine ************************************************/
/* ****************************************************************** */


int main(int argc, char* argv[]) {
	
	constexpr const int L = 14; // number of sites
	constexpr const int d = 2; // dim. of state space  
	int lda = 1<<(L-1);

	hipDoubleComplex *d_Psi = nullptr;
	double *d_S = nullptr;
	hipDoubleComplex *d_SCC = nullptr; // hipDoubleComplex version of S, 1-dim. array of singular values
	hipDoubleComplex *d_U = nullptr; 
	hipDoubleComplex *d_VT = nullptr; 
	int *devInfo = NULL;
	hipDoubleComplex *d_work = nullptr;
	double *d_rwork = NULL;

	// cuBLAS matrix multiplication step
	hipDoubleComplex *d_US = nullptr;  
	
	// calculate new Psi
	hipDoubleComplex *d_Psi_new = nullptr;


	int lwork =0;

	/* ************************************************************** */
	/* ************************************************************** */
	/* ****** BOILERPLATE initialization, values ******************** */
	/* ************************************************************** */
//	create_fixed_CC_mat(d,L,Psi);
//	printMatrix(lda,d, &Psi.data() ,lda,"Psi");
//	hipDoubleComplex Psi[lda*d]; 
//	hipDoubleComplex*  Psi = new hipDoubleComplex[lda*d]; 
	// boilerplate, initialization; matrix assumed to be column-major ordering 
	std::array<hipDoubleComplex, (1<<(L-1))*d> Psi;
	{
		int M = 1<<(L-1);  // d^{(L-1)}, where d is dim. of state space, L is number of sites
		for (int i =0; i< M; i++) { // i is the "row" of a matrix, it's an index
			double f = ((double) i*(0.9/M)+0.1);
			double theta_f = 2.* acos( -1.)*f;
			hipDoubleComplex Ad0 = { f*cos(theta_f) ,  f*sin(theta_f) } ; 
			hipDoubleComplex Ad1 = { (1.-f)*sin(theta_f) , (1.-f)*cos(theta_f) } ; 
			Psi[i + M*0] = Ad0 ; 
			Psi[i + M*1] = Ad1 ; 
			std::cout << std::setprecision(5) << Ad0.x << "+i" << Ad0.y << " " << Ad1.x << "+i" << Ad1.y << std::endl; 
		}
	}


	GpuTimer timer;
	/* ************************************************************** */
	/* ****** END of BOILERPLATE initialization, values ************* */
	/* ************************************************************** */

	hipsolverHandle_t cusolverH = nullptr;
	hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
// step 1: create cusolverDn handle 
	cusolver_status = hipsolverDnCreate(&cusolverH);

	hipblasHandle_t cublasH = nullptr;
	hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
// step 1b: create cublas handle 
	hipblasCreate(&cublasH);

// step 2: copy A and B to device
	hipMalloc((void**)&d_Psi , sizeof(hipDoubleComplex)*lda*d);
	hipMalloc((void**)&d_S , sizeof(double)*d);
	hipMalloc((void**)&d_SCC , sizeof(hipDoubleComplex)*d);
	hipMalloc((void**)&d_U , sizeof(hipDoubleComplex)*lda*lda);
	hipMalloc((void**)&d_VT , sizeof(hipDoubleComplex)*d*d);
	hipMalloc((void**)&devInfo, sizeof(int));

	hipMalloc((void**)&d_US , sizeof(hipDoubleComplex)*lda*lda);

	hipMalloc((void**)&d_Psi_new , sizeof(hipDoubleComplex)*lda*d);


	hipMemcpy(d_Psi, Psi.data(), sizeof(hipDoubleComplex)*lda*d, hipMemcpyHostToDevice);

// step 3: query working space of SVD 
	timer.Start();  // timer "boilerplate"

	cusolver_status = hipsolverDnZgesvd_bufferSize(cusolverH,  // cusolver Handle
		lda,d,  // matrix size dimensions of Psi
		&lwork );
	hipMalloc((void**)&d_work , sizeof(hipDoubleComplex)*lwork);

// step 4: compute SVD 
	cusolver_status = hipsolverDnZgesvd(cusolverH,'A','A',
		lda,d,
		d_Psi,lda,
		d_S,
		d_U,lda, 	// ldu
		d_VT,d, 	// ldvt,
		d_work,lwork,d_rwork,devInfo);

// change type of 1-dim. array of singular values S from double to hipDoubleComplex
	hipMemcpy2D(d_SCC, // dst - Destination memory address
					sizeof(hipDoubleComplex), // dpitch - Pitch of destination memory (1 hipDoubleComplex, so skip over 2 double values)
					d_S, 	// src
					1*sizeof(double), 	// spitch
					sizeof(double), 	// width of matrix transfer (columns in bytes)
					d, 					// height of matrix transfer (rows)
					hipMemcpyDeviceToDevice); 

// Matrix Multiply U*S to obtain US, or new Psi, "flattened".  
	cublas_status = hipblasZdgmm(cublasH, HIPBLAS_SIDE_RIGHT,
		lda,lda,
		d_U,lda,
		d_SCC,1,
		d_US,lda);

// obtain new Psi, 1st step - "reduce" matrix size dim. to the Schmidt rank  
	hipMemcpy(d_Psi_new, d_US, sizeof(hipDoubleComplex)*lda*d, hipMemcpyDeviceToDevice);

// sanity check  
//	hipDoubleComplex US[lda*lda]; // d^{L-1)-by-d unitary matrix
	hipDoubleComplex VT[d*d]; // d-by-d unitary matrix
	hipMemcpy(VT,d_VT, sizeof(hipDoubleComplex)*d*d,hipMemcpyDeviceToHost); 
//	hipMemcpy(US,d_US, sizeof(hipDoubleComplex)*lda*lda,hipMemcpyDeviceToHost); 	
//	printMatrix(lda, lda, US, lda, "US"); // 1 should observe many 0 entries which is expected
	printMatrix(d, d, VT, d, "VT");
//	hipDoubleComplex Psi_new[lda*d]; // d^{L-1)-by-d unitary matrix
//	hipMemcpy(Psi_new,d_Psi_new, sizeof(hipDoubleComplex)*lda*d,hipMemcpyDeviceToHost); 	
//	printMatrix(lda, d, Psi_new, lda, "Psi_new");




// free resources
	if (d_Psi		) hipFree(d_Psi);
	if (d_S		) hipFree(d_S);
	if (d_SCC		) hipFree(d_SCC);

	if (d_U 	) hipFree(d_U);
	if (d_US	) hipFree(d_US);
	if (d_work	) hipFree(d_work);


/* ****************************************************************** */
/* ****** 2nd iteration ********************************************* */
/* ****************************************************************** */
 
	double *d_S_l2 = nullptr;
	hipDoubleComplex *d_SCC_l2 = nullptr; // hipDoubleComplex version of S, 1-dim. array of singular values
	hipDoubleComplex *d_U_l2 = nullptr; 
	hipDoubleComplex *d_VT_l2 = nullptr; 
	hipDoubleComplex *d_work_l2 = nullptr;

	// cuBLAS matrix multiplication step
	hipDoubleComplex *d_US_l2 = nullptr;  
	
	// calculate new Psi
	hipDoubleComplex *d_Psi_new_l2 = nullptr;

// step 2: device memory (GPU) allocation
	lda = (1<<(L-2)); 
	const int dr = d*d;

// sanity check
//	hipDoubleComplex Psi_new[lda*dr];  
//	hipMemcpy(Psi_new,d_Psi_new, sizeof(hipDoubleComplex)*lda*lda,hipMemcpyDeviceToHost); 
//	printMatrix(lda, lda, Psi_new, lda, "Psi_new");



	hipMalloc((void**)&d_S_l2 , sizeof(double)*dr*dr);
	hipMalloc((void**)&d_SCC_l2 , sizeof(hipDoubleComplex)*dr*dr);
	hipMalloc((void**)&d_U_l2 , sizeof(hipDoubleComplex)*lda*lda);
	hipMalloc((void**)&d_VT_l2 , sizeof(hipDoubleComplex)*dr*dr);

	hipMalloc((void**)&d_US_l2 , sizeof(hipDoubleComplex)*lda*lda);

	hipMalloc((void**)&d_Psi_new_l2 , sizeof(hipDoubleComplex)*lda*dr);

// step 3: query working space of SVD 
	cusolver_status = hipsolverDnZgesvd_bufferSize(cusolverH,  // cusolver Handle
		lda,dr,  // matrix size dimensions of Psi
		&lwork );
	hipMalloc((void**)&d_work_l2 , sizeof(hipDoubleComplex)*lwork);

// step 4: compute SVD 
	cusolver_status = hipsolverDnZgesvd(cusolverH,'A','A',
		lda,dr,
		d_Psi_new,lda,
		d_S_l2,
		d_U_l2,lda, 	// ldu
		d_VT_l2,dr, 	// ldvt,
		d_work_l2,lwork,NULL,devInfo);

// change type of 1-dim. array of singular values S from double to hipDoubleComplex
	hipMemcpy2D(d_SCC_l2, // dst - Destination memory address
					sizeof(hipDoubleComplex), // dpitch - Pitch of destination memory (1 hipDoubleComplex, so skip over 2 double values)
					d_S_l2, 	// src
					1*sizeof(double), 	// spitch
					sizeof(double), 	// width of matrix transfer (columns in bytes)
					dr, 					// height of matrix transfer (rows)
					hipMemcpyDeviceToDevice); 

// sanity check
//	hipDoubleComplex S_l2[dr]; // 1-dim. array 
//	hipDoubleComplex U_l2[lda*lda];  
//	hipMemcpy(S_l2,d_SCC_l2, sizeof(hipDoubleComplex)*dr,hipMemcpyDeviceToHost); 
//	hipMemcpy(U_l2,d_U_l2, sizeof(hipDoubleComplex)*lda*lda,hipMemcpyDeviceToHost); 
//	print1darr<hipDoubleComplex>(dr,S_l2,1," S_l2 ");
//	printMatrix(lda, lda, U_l2, lda, "U_l2");

	
// Matrix Multiply U*S to obtain US, or new Psi, "flattened".  
	cublas_status = hipblasZdgmm(cublasH, HIPBLAS_SIDE_RIGHT,
		lda,lda,
		d_U_l2,lda,
		d_SCC_l2,1,
		d_US_l2,lda);

// obtain new Psi, 1st step - "reduce" matrix size dim. to the Schmidt rank  
	hipMemcpy(d_Psi_new_l2, d_US_l2, sizeof(hipDoubleComplex)*lda*dr, hipMemcpyDeviceToDevice);


	// timer "boilerplate"
	timer.Stop();


	hipDeviceSynchronize();

// sanity check  
//	hipDoubleComplex US_l2[lda*lda]; // d^{L-2)-by-dr unitary matrix
//	hipMemcpy(US_l2,d_US_l2, sizeof(hipDoubleComplex)*lda*lda,hipMemcpyDeviceToHost); 	
//	printMatrix(lda, lda, US_l2, lda, "US_l2 (2nd iteration)");
//	hipDoubleComplex Psi_new_l2[lda*dr]; // d^{L-1)-by-d unitary matrix
	std::array<hipDoubleComplex, (1<<(L-2))*dr> Psi_new_l2;
	hipMemcpy(Psi_new_l2.data(),d_Psi_new_l2, sizeof(hipDoubleComplex)*lda*dr,hipMemcpyDeviceToHost); 	
	{
	for (int row =0; row <lda; row++) {
		for (int col =0 ; col <dr ; col++) {
			hipDoubleComplex Areg = Psi_new_l2[row + col*lda]; 
			std::cout << std::setprecision(5) << Areg.x << "+i" << Areg.y << " " ; 
		}
		std::cout << std::endl;
	}
}
//	hipDeviceSynchronize();
//	printMatrix(lda, dr, Psi_new_l2, lda, "Psi_new_l2 (2nd iteration)");
//	std::cout << " lda for 2nd iteration : " << lda << std::endl; // sanity check
//	std::cout << " dr for 2nd iteration : " << dr << std::endl; // sanity check for matrix size dim.

	hipDoubleComplex VT_l2[dr*dr]; // d^{L-1)-by-d unitary matrix
	hipMemcpy(VT_l2,d_VT_l2, sizeof(hipDoubleComplex)*dr*dr,hipMemcpyDeviceToHost); 	
	std::cout << "\n VT = (matlab base-1), 2nd. iteration " << std::endl; 
	printMatrix(dr, dr, VT_l2, dr, "VT 2nd iteration");
	std::cout << "===== " << std::endl;
	

//	std::cout << " Calculation of 2 iterations of SVD and matrix multiplication  : " << std::setprecision(7) << timer.Elapsed() << " ms " << 
	std::cout << " Calculation of 2 iterations of SVD and matrix multiplication  : " << timer.Elapsed() << " ms " << 
		" for " << (1<<L) << " states (of the system " << std::endl;


// free all resources
	if (d_VT	) hipFree(d_VT);
	if (d_VT_l2	) hipFree(d_VT_l2);

	if (d_U_l2	) hipFree(d_U_l2);
	if (d_work_l2	) hipFree(d_work_l2);

	if (devInfo ) hipFree(devInfo);

	if (d_rwork	) hipFree(d_rwork);
	
	if (d_US_l2	) hipFree(d_US_l2);
	if (d_Psi_new		) hipFree(d_Psi_new);
	if (d_Psi_new_l2		) hipFree(d_Psi_new_l2);

		
	if (cusolverH) hipsolverDnDestroy(cusolverH);
	if (cublasH	) hipblasDestroy(cublasH);
	
	hipDeviceReset();
	return 0;

}
