#include "hip/hip_runtime.h"
/* dev_R2grid.cu
 * R3 under discretization (discretize functor) to a (staggered) grid
 * Ernest Yeung  ernestyalumni@gmail.com
 * 2016115
 * 
 * compilation tip: (compile separately)
 * nvcc -std=c++11 -c ./physlib/dev_R2grid.cu -o dev_R2grid.o
 * 
 */
#include "dev_R2grid.h"

//__constant__ int dev_Ld[2];

// constructor
__host__ Dev_Grid2d::Dev_Grid2d( dim3 Ld_in, const int NDIR) : Ld(Ld_in), NDIR(NDIR)
{
	staggered_Ld.x  = Ld.x+2;
	staggered_Ld.y  = Ld.y+2;

	checkCudaErrors( 
		hipMalloc((void **)&this->f, this->NFLAT() * NDIR * sizeof(float)) );
	checkCudaErrors( 
		hipMalloc((void **)&this->feq, this->NFLAT() * NDIR * sizeof(float)) );
	checkCudaErrors( 
		hipMalloc((void **)&this->f_new, this->NFLAT() * NDIR * sizeof(float)) );
	
	checkCudaErrors( 
		hipMalloc((void **)&this->rh, this->NFLAT() * sizeof(float)) );
	checkCudaErrors( 
		hipMalloc((void **)&this->u, this->NFLAT() * sizeof(float2)) );

	// set to 0
	checkCudaErrors( 
		hipMemset(this->f, 0, this->NFLAT() * NDIR * sizeof(float)) );
	checkCudaErrors( 
		hipMemset(this->feq, 0,this->NFLAT() * NDIR * sizeof(float)) );
	checkCudaErrors( 
		hipMemset(this->f_new, 0,this->NFLAT() * NDIR * sizeof(float)) );

	checkCudaErrors( 
		hipMemset(this->rh, 0,this->NFLAT() * sizeof(float)) );
	checkCudaErrors( 
		hipMemset(this->u, 0,this->NFLAT() * sizeof(float2)) );

	
	
	
}

// destructor

__host__ Dev_Grid2d::~Dev_Grid2d() {

	// REMOVE this destructor (i.e.comment it out) when you want to use OpenGL graphics
	
	checkCudaErrors(
		hipFree( this->f ));
	checkCudaErrors(
		hipFree( this->feq ));
	checkCudaErrors(
		hipFree( this->f_new ));

	checkCudaErrors(
		hipFree( this->rh ));
	checkCudaErrors(
		hipFree( this->u ));


}


__host__ int Dev_Grid2d :: NFLAT() {
	return Ld.x*Ld.y;
}	

__host__ int Dev_Grid2d :: staggered_SIZE() {
	return (staggered_Ld.x)*(staggered_Ld.y);
}	

__host__ int Dev_Grid2d :: flatten(const int i_x, const int i_y ) {
	return i_x+i_y*Ld.x  ;
}

__host__ int Dev_Grid2d :: staggered_flatten(const int i_x, const int i_y ) {
	return i_x+i_y*(staggered_Ld.x)  ;
}

