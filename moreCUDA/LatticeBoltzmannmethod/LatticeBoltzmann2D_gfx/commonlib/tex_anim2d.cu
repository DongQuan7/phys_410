#include "hip/hip_runtime.h"
/* tex_anim2d.cu
 * 2-dim. GPU texture animation 
 * Ernest Yeung  ernestyalumni@gmail.com
 * 20160720
 */

	// ################################################################
	// MANUALLY change minval, maxval in 
	// __global__ void float_to_char 
	// ################################################################

#include "tex_anim2d.h"
  
int iterationCount = 0 ;


// interactions

void keyboard_func( unsigned char key, int x, int y) {

	if (key==27) {
//		std::exit(0) ;
		exit(0);
	}
	glutPostRedisplay();
}
	
void mouse_func( int button, int state, int x, int y ) {
	glutPostRedisplay();
}

void idle() {
	++iterationCount;
	glutPostRedisplay();
}

void printInstructions() {
	printf("2 dim. texture animation \n"

			"Exit                           : Esc\n"
	
	);
}

// make* functions make functions to pass into OpenGL (note OpenGL is inherently a C API
void make_draw_texture(int w, int h) {
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, w, h, 0, GL_RGBA, 
		GL_UNSIGNED_BYTE, NULL);
	glEnable(GL_TEXTURE_2D);

	glBegin(GL_QUADS);
	glTexCoord2f(0.0f, 0.0f); glVertex2f(0,0);
	glTexCoord2f(0.0f, 1.0f); glVertex2f(0,h);
	glTexCoord2f(1.0f, 1.0f); glVertex2f(w,h);
	glTexCoord2f(1.0f, 0.0f); glVertex2f(w,0);  // glVertex2f(float(w),0);
	glEnd();
	
	glDisable(GL_TEXTURE_2D);
}	



__global__ void float_to_char( uchar4* dev_out, const float* outSrc, const int L_x, const int L_y) {
	const int k_x = threadIdx.x + blockDim.x * blockIdx.x ;
	const int k_y = threadIdx.y + blockDim.y * blockIdx.y ;
	
	const int k = k_x + k_y * blockDim.x * gridDim.x;
	if ((k_x >= L_x) || (k_y >= L_y)) {
		return ; }
		

		
	dev_out[k].x = 0;
	dev_out[k].z = 0;
	dev_out[k].y = 0;
	dev_out[k].w = 255;
	
	float value = outSrc[k] ; 

	// convert to long rainbox RGB*
	// 1. convert to [0.0,1.0] scale from [minval,maxval] (set minval,maxval MANUALLY)
	// MANUALLY change minval, maxval
	const float minval = 2.55f;
	const float maxval = 2.85f;

	value = (value - minval) / ( maxval - minval) ; 
	if (value < minval ) { value = 0.0; }
	else if (value > maxval ) { value = 1.0; }
	
	// 2. convert to long rainbox RGB*
	value = value / 0.20;
	int valueint  = ((int) floorf( value )); // this is the integer part
	int valuefrac = ((int) floorf( 255*(value - valueint)) );
	
	switch( valueint )
	{
		case 0: dev_out[k].x = 255; dev_out[k].y = valuefrac; dev_out[k].z = 0;
		dev_out[k].w = 255;
		break;
		case 1: dev_out[k].x = 255- valuefrac; dev_out[k].y = 255; dev_out[k].z = 0;
		dev_out[k].w = 255;
		break;
		case 2: dev_out[k].x = 0; dev_out[k].y = 255; dev_out[k].z = valuefrac;
		dev_out[k].w = 255;
		break;
		case 3: dev_out[k].x = 0; dev_out[k].y = 255- valuefrac; dev_out[k].z = 255;
		dev_out[k].w = 255;
		break;
		case 4: dev_out[k].x = valuefrac; dev_out[k].y = 0; dev_out[k].z = 255;
		dev_out[k].w = 255;
		break;
		case 5: dev_out[k].x = 255; dev_out[k].y = 0; dev_out[k].z = 255;
		dev_out[k].w = 255;
		break;
	}
}

// float2_to_char, input float2 * velocity array, pick out .x component, transform to a char
__global__ void float2_to_char( uchar4* dev_out, const float2* outSrc, const int L_x, const int L_y) {
	const int k_x = threadIdx.x + blockDim.x * blockIdx.x ;
	const int k_y = threadIdx.y + blockDim.y * blockIdx.y ;
	
	const int k = k_x + k_y * blockDim.x * gridDim.x;
	if ((k_x >= L_x) || (k_y >= L_y)) {
		return ; }
		

		
	dev_out[k].x = 0;
	dev_out[k].z = 0;
	dev_out[k].y = 0;
	dev_out[k].w = 255;
	
	float value = outSrc[k].x ; 

	// convert to long rainbox RGB*
	// 1. convert to [0.0,1.0] scale from [minval,maxval] (set minval,maxval MANUALLY)
	// MANUALLY change minval, maxval
	const float minval = -0.5f;
	const float maxval = 0.500f;

	value = (value - minval) / ( maxval - minval) ; 
	if (value < minval ) { value = 0.0; }
	else if (value > maxval ) { value = 1.0; }
	
	// 2. convert to long rainbox RGB*
	value = value / 0.20;
	int valueint  = ((int) floorf( value )); // this is the integer part
	int valuefrac = ((int) floorf( 255*(value - valueint)) );
	
	switch( valueint )
	{
		case 0: dev_out[k].x = 255; dev_out[k].y = valuefrac; dev_out[k].z = 0;
		dev_out[k].w = 255;
		break;
		case 1: dev_out[k].x = 255- valuefrac; dev_out[k].y = 255; dev_out[k].z = 0;
		dev_out[k].w = 255;
		break;
		case 2: dev_out[k].x = 0; dev_out[k].y = 255; dev_out[k].z = valuefrac;
		dev_out[k].w = 255;
		break;
		case 3: dev_out[k].x = 0; dev_out[k].y = 255- valuefrac; dev_out[k].z = 255;
		dev_out[k].w = 255;
		break;
		case 4: dev_out[k].x = valuefrac; dev_out[k].y = 0; dev_out[k].z = 255;
		dev_out[k].w = 255;
		break;
		case 5: dev_out[k].x = 255; dev_out[k].y = 0; dev_out[k].z = 255;
		dev_out[k].w = 255;
		break;
	}
}

// from physical scalar values to color intensities on an OpenGL bitmap
__global__ void floatux_to_char( uchar4* dev_out, hipSurfaceObject_t uSurf, 
									const int L_x, const int L_y) {
	const int k_x = threadIdx.x + blockDim.x * blockIdx.x ; 
	const int k_y = threadIdx.y + blockDim.y * blockIdx.y ; 
	
	const int k = k_x + k_y * blockDim.x *gridDim.x ; 
	if ((k_x >= L_x ) || (k_y >= L_y)) {
		return ; }
	
	dev_out[k].x = 0;
	dev_out[k].y = 0;
	dev_out[k].z = 0;
	dev_out[k].w = 255;
	 
	float2 tempu; 
	surf2Dread(&tempu, uSurf, k_x * 8, k_y );

	// clipping part
	const float scale = 2.f ;
	const float newval = tempu.x / scale; 
	 
	int n = 256 * newval ; 
	n = max( min( n, 255) , 0 ) ; 
	// END of clipping part
	 
	const unsigned char intensity = n ; 
	dev_out[k].x = intensity ;  // higher magnitude -> more red
	dev_out[k].z = 255 - intensity ; // lower magnitude -> more blue
}
