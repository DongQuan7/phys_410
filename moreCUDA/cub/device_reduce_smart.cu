/**
 * @file   : device_reduce_smart.cu
 * @brief  : demonstrating usage of CUDA CUB's device-(wide) reduce or sum, also with 
 * 			 C++11/14 smart pointers
 * @author : Ernest Yeung <ernestyalumni@gmail.com>
 * @date   : 20170902  
 * @ref    :  
 * 
 * If you find this code useful, feel free to donate directly and easily at this direct PayPal link: 
 * 
 * https://www.paypal.com/cgi-bin/webscr?cmd=_donations&business=ernestsaveschristmas%2bpaypal%40gmail%2ecom&lc=US&item_name=ernestyalumni&currency_code=USD&bn=PP%2dDonationsBF%3abtn_donateCC_LG%2egif%3aNonHosted 
 * 
 * which won't go through a 3rd. party such as indiegogo, kickstarter, patreon.  
 * Otherwise, I receive emails and messages on how all my (free) material on 
 * physics, math, and engineering have helped students with their studies, 
 * and I know what it's like to not have money as a student, but love physics 
 * (or math, sciences, etc.), so I am committed to keeping all my material 
 * open-source and free, whether or not 
 * sufficiently crowdfunded, under the open-source MIT license: 
 * 	feel free to copy, edit, paste, make your own versions, share, use as you wish.  
 *  Just don't be an asshole and not give credit where credit is due.  
 * Peace out, never give up! -EY
 * 
 * */
/* 
 * COMPILATION TIP
 * nvcc -std=c++11 -I ~/cub/cub-1.7.3 device_reduce_smart.cu -o device_reduce_smart.exe
 * 
 * */

#include <iostream>	// std::cout
#include <vector> 	// std::vector
#include <memory> 	// std::unique_ptr


#include <hipcub/hipcub.hpp>


//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------
int main(int argc, char* argv[]) {
	constexpr const int Lx = (1 << 8);
	std::cout << " Lx : " << Lx << std::endl;

	// Allocate host arrays
	std::vector<float> f_vec(Lx,1.f);
	
	// Allocate problem device arrays
	auto deleter=[&](float* ptr){ hipFree(ptr); };
//	std::unique_ptr<float[], decltype(deleter)> d_in(new float[Lx], deleter);
	std::shared_ptr<float> d_in(new float[Lx], deleter);
	hipMalloc((void **) &d_in, Lx * sizeof(float));


    // Initialize device input
	hipMemcpy(d_in.get(), f_vec.data(), Lx*sizeof(float),hipMemcpyHostToDevice);

	// Allocate device output array
//	std::unique_ptr<float, decltype(deleter)> d_out(new float(0.f), deleter);
	std::shared_ptr<float> d_out(new float(0.f), deleter);
	hipMalloc((void **) &d_out, 1 * sizeof(float));


    // Request and allocate temporary storage
//	std::unique_ptr<void, decltype(deleter)> d_temp_storage(nullptr, deleter);
	std::shared_ptr<void> d_temp_storage(nullptr, deleter);
//	void* d_temp_storage = nullptr;
	
	size_t 		temp_storage_bytes = 0;

	hipcub::DeviceReduce::Sum( d_temp_storage.get(), temp_storage_bytes, d_in.get(),d_out.get(),Lx);

//	hipMalloc( (void **) d_temp_storage.get(), temp_storage_bytes);
	hipMalloc((void **) &d_temp_storage, temp_storage_bytes);
	
	// Run
	hipcub::DeviceReduce::Sum(d_temp_storage.get(),temp_storage_bytes,d_in.get(),d_out.get(),Lx);

	// Allocate output host array
	std::vector<float> g_vec(1,0.f);
	
	// Copy results from Device to Host
	hipMemcpy(g_vec.data(), d_out.get(), 1*sizeof(float),hipMemcpyDeviceToHost);

	// print out result:
	std::cout << " g_vec[0] : " << g_vec[0] << std::endl;

	// Clean up
//	hipFree(d_temp_storage);
	hipDeviceReset();
	return 0;
}
