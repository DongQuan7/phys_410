#include "hip/hip_runtime.h"
/* dd_1d_global.cu
 * 1-dimensional double derivative (dd for '') by finite difference with global memory
 * Ernest Yeung  ernestyalumni@gmail.com
 * 20160625
 */
#include "dd_1d_global.h"
#define M_x 64 // number of threads per block in x-direction

__global__ void ddKernel(float *d_out, const float *d_in, int L_x, float h) {
	const int k_x = threadIdx.x + blockDim.x*blockIdx.x;
	if (k_x >= L_x) return;
	d_out[k_x] = (d_in[k_x-1]-2.f*d_in[k_x]+d_in[k_x+1])/(h*h);
}

void ddParallel(float *out, const float *in, int n, float h) {
	float *d_in = 0, *d_out = 0;
	
	hipMalloc(&d_in, n*sizeof(float));
	hipMalloc(&d_out, n*sizeof(float));
	hipMemcpy(d_in, in, n*sizeof(float), hipMemcpyHostToDevice);
	
	ddKernel<<<(n + M_x - 1)/M_x, M_x>>>(d_out, d_in, n , h);
	
	hipMemcpy(out, d_out, n*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_in);
	hipFree(d_out);
}

