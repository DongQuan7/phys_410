/* dist_3d.cu
 *  
 * based on code from CUDA for Engineers (cudaforengineers)
 * Ernest Yeung  ernestyalumni@gmail.com
 * 20160625
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#define Nthreads_x 32 // total number of threads in x-direction
#define Nthreads_y 32 // total number of threads in y-direction
#define Nthreads_z 32 // total number of threads in z-direction

#define M_x 8 // number of threads per block in x-direction
#define M_y 8 // number of threads per block in y-direction
#define M_z 8 // number of threads per block in z-direction

int blocksNeeded(int N_i, int M_i) { return (N_i+M_i-1)/M_i; }

__device__ float distance(int k_x, int k_y, int k_z, float3 x_0) {
	return sqrtf((k_x - x_0.x)*(k_x - x_0.x) + (k_y - x_0.y)*(k_y - x_0.y) + 
					(k_z - x_0.z)*(k_z - x_0.z));
	}

__global__ void distance(float *d_out, int L_x, int L_y, int L_z, float3 x_0) {
	// map from threadIdx/blockIdx to (k_x,k_y,k_z) grid position
	const int k_x = threadIdx.x + blockIdx.x*blockDim.x;
	const int k_y = threadIdx.y + blockIdx.y*blockDim.y;
	const int k_z = threadIdx.z + blockIdx.z*blockDim.z;
	const int offset = k_x + k_y*L_x + k_z*L_x*L_y;
	if ((k_x >= L_x) || (k_y >= L_y) || (k_z >= L_z)) return;
	d_out[offset] = distance( k_x, k_y, k_z, x_0); // compute and store result
}
	
int main() {
	float *out = (float *)malloc(Nthreads_x*Nthreads_y*Nthreads_z*sizeof(float));
	float *d_out = 0;
	hipMalloc(&d_out, Nthreads_x*Nthreads_y*Nthreads_z*sizeof(float));
	
	const float3 x_0 = { 0.0f, 0.0f, 0.0f }; // set reference position x_0
	const dim3 blockSize( M_x, M_y, M_z);
	const dim3 gridSize( blocksNeeded(Nthreads_x, M_x), blocksNeeded(Nthreads_y, M_y), 
						 blocksNeeded(Nthreads_z, M_z));
	distance<<<gridSize,blockSize>>>(d_out, Nthreads_x, Nthreads_y, Nthreads_z, x_0);
	hipMemcpy(out, d_out, Nthreads_x*Nthreads_y*Nthreads_z*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_out);

	// sanity check
	int testx = (int) 7*Nthreads_x*Nthreads_y*Nthreads_z/10;
	printf("At %d the distance is %f \n", testx, out[testx]); 

	free(out);
	return 0;
}

