#include "hip/hip_runtime.h"
/* dist_3db.cu
 * 3-dim. Euclidean distance
 * Ernest Yeung  ernestyalumni@gmail.com
 * 20160625
 */
#include <stdio.h>
#include "../../common/errors.h"
#define Nthreads_x 1000 // total number of threads in x-direction (304 works)
#define Nthreads_y 1000 // total number of threads in y-direction (304 works)
#define Nthreads_z 1000 // total number of threads in z-direction (304 works)

#define M_x 20 // number of threads per block in x-direction
#define M_y 20 // number of threads per block in y-direction
#define M_z 20 // number of threads per block in z-direction

int blocksNeeded(int N_i, int M_i) { return (N_i+M_i-1)/M_i; }

__device__ float distance(int k_x, int k_y, int k_z, float3 x_0) {
	return sqrtf((k_x - x_0.x)*(k_x - x_0.x) + (k_y - x_0.y)*(k_y - x_0.y) + 
					(k_z - x_0.z)*(k_z - x_0.z));
	}

__global__ void distance(float *d_out, int L_x, int L_y, int L_z, float3 x_0) {
	// sanity check
//	printf("Hello thread x=%d,y=%d,z=%d \n", threadIdx.x, threadIdx.y,threadIdx.z);
	
	// map from threadIdx/blockIdx to (k_x,k_y,k_z) grid position
	const int k_x = threadIdx.x + blockIdx.x*blockDim.x;
	const int k_y = threadIdx.y + blockIdx.y*blockDim.y;
	const int k_z = threadIdx.z + blockIdx.z*blockDim.z;
	const int offset = k_x + k_y*L_x + k_z*L_x*L_y;
	if ((k_x >= L_x) || (k_y >= L_y) || (k_z >= L_z)) return;
	d_out[offset] = distance( k_x, k_y, k_z, x_0); // compute and store result

	// sanity check
//	printf("On global thread index x=%d,y=%d,z=%d, distance=%f\n",
//			k_x,k_y,k_z,d_out[offset]);
}
	
int main() {
	float *out = (float *)malloc(Nthreads_x*Nthreads_y*Nthreads_z*sizeof(float));
	float *d_out = 0;
	hipMalloc(&d_out, Nthreads_x*Nthreads_y*Nthreads_z*sizeof(float));
	
	const float3 x_0 = { 0.0f, 0.0f, 0.0f }; // set reference position x_0
	const dim3 blockSize( M_x, M_y, M_z);
	const dim3 gridSize( blocksNeeded(Nthreads_x, M_x), blocksNeeded(Nthreads_y, M_y), 
						 blocksNeeded(Nthreads_z, M_z));
	distance<<<gridSize,blockSize>>>(d_out, Nthreads_x, Nthreads_y, Nthreads_z, x_0);
//	hipDeviceSynchronize();
	hipMemcpy(out, d_out, Nthreads_x*Nthreads_y*Nthreads_z*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_out);

	// sanity check
	int testx = (int) 9.*Nthreads_x/10.;
	int testy = (int) 9.*Nthreads_y/10.;
	int testz = (int) 9.*Nthreads_z/10.;
	printf("At (%d,%d,%d), the distance is %f \n", testx, testy,testz,
			out[testx+testy*Nthreads_x+testz*Nthreads_x*Nthreads_y]); 

	free(out);
	return 0;
}
