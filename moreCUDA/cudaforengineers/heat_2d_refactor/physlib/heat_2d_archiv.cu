#include "hip/hip_runtime.h"
/* heat_2d.cu
 * 2-dim. Laplace eq. (heat eq.) by finite difference with shared memory
 * Ernest Yeung  ernestyalumni@gmail.com
 * 20160625
 */
#include "heat_2d.h"

#define RAD 1 // radius of the stencil; helps to deal with "boundary conditions" at (thread) block's ends

__constant__ float dev_Deltat[1];

__constant__ float dev_heat_params[2];



int blocksNeeded( int N_i, int M_i) { return (N_i+M_i-1)/M_i; }

__device__ unsigned char clip(int n) { return n > 255 ? 255 : (n < 0 ? 0 : n);}

__device__ int idxClip( int idx, int idxMax) {
	return idx > (idxMax - 1) ? (idxMax - 1): (idx < 0 ? 0 : idx);
}

__device__ int flatten(int col, int row, int width, int height) {
	return idxClip(col, width) + idxClip(row,height)*width;
}

__global__ void resetKernel(float *d_temp, int w, int h, BC bc) {
	const int col = blockIdx.x*blockDim.x + threadIdx.x;
	const int row = blockIdx.y*blockDim.y + threadIdx.y;
	if ((col >= w) || (row >= h)) return;
	d_temp[row*w + col] = bc.t_a;
}


__global__ void tempKernel(float *d_temp, int w, int h, BC bc) {
	extern __shared__ float s_in[];
	// global indices
	const int col = threadIdx.x + blockDim.x * blockIdx.x;
	const int row = threadIdx.y + blockDim.y * blockIdx.y;
	if ((col >= w ) || (row >= h )) return;
	const int idx = flatten(col, row, w, h);
	// local width and height
	const int s_w = blockDim.x + 2 * RAD;
	const int s_h = blockDim.y + 2 * RAD;
	// local indices
	const int s_col = threadIdx.x + RAD;
	const int s_row = threadIdx.y + RAD;
	const int s_idx = flatten(s_col, s_row, s_w, s_h);
	// assign default color values for d_out (black)

	// Load regular cells
	s_in[s_idx] = d_temp[idx];
	// Load halo cells
	if (threadIdx.x < RAD ) {
		s_in[flatten(s_col - RAD, s_row, s_w, s_h)] = d_temp[flatten(col - RAD, row, w, h)];
		s_in[flatten(s_col + blockDim.x, s_row, s_w, s_h)] = d_temp[flatten(col + blockDim.x, row, w, h)];
	}
	if (threadIdx.y < RAD) {
		s_in[flatten(s_col, s_row - RAD, s_w, s_h)] = d_temp[flatten(col, row - RAD, w, h)];
		s_in[flatten(s_col, s_row + blockDim.y, s_w, s_h)] = d_temp[flatten(col, row + blockDim.y, w, h)];
	}
	
	// Calculate squared distance from pipe center
	float dSq = ((col - bc.x)*(col - bc.x) + (row - bc.y)*(row - bc.y));
	// If inside pipe, set temp to t_s and return
	if (dSq < bc.rad*bc.rad) {
		d_temp[idx] = bc.t_s;
		return;
	}
	// If outside plate, set temp to t_a and return
	if ((col == 0 ) || (col == w - 1) || (row == 0 ) ||
		(col + row < bc.chamfer) || (col - row > w - bc.chamfer)) {
			d_temp[idx] = bc.t_a;
			return;
	}
	// If point is below ground, set temp to t_g and return
	if (row == h - 1) {
		d_temp[idx] = bc.t_g;
		return;
	}
	__syncthreads();
	// For all the remaining points, find temperature and set colors.
	float temp = 0.25f*(s_in[flatten(s_col - 1, s_row, s_w, s_h)] + 
				 s_in[flatten(s_col + 1,s_row,s_w,s_h)] + 
				 s_in[flatten(s_col, s_row - 1,s_w, s_h)] + 
				 s_in[flatten(s_col, s_row + 1, s_w, s_h)]);
	d_temp[idx] = temp;

}

__global__ void tempKernel2(float *d_temp, BC bc) {
	extern __shared__ float s_in[];
	// global indices
	const int k_x = threadIdx.x + blockDim.x * blockIdx.x;
	const int k_y = threadIdx.y + blockDim.y * blockIdx.y;
	if ((k_x >= dev_Ld[0] ) || (k_y >= dev_Ld[1] )) return;
	const int k = flatten(k_x, k_y, dev_Ld[0], dev_Ld[1]);
	// local width and height
	const int S_x = blockDim.x + 2 * RAD;
	const int S_y = blockDim.y + 2 * RAD;
	// local indices
	const int s_x = threadIdx.x + RAD;
	const int s_y = threadIdx.y + RAD;
	const int s_k = flatten(s_x, s_y, S_x, S_y);
	// assign default color values for d_out (black)

	// Load regular cells
	s_in[s_k] = d_temp[k];
	// Load halo cells
	if (threadIdx.x < RAD ) {
		s_in[flatten(s_x - RAD, s_y, S_x, S_y)] = d_temp[flatten(k_x - RAD, k_y, dev_Ld[0], dev_Ld[1])];
		s_in[flatten(s_x + blockDim.x, s_y, S_x, S_y)] = d_temp[flatten(k_x + blockDim.x, k_y, dev_Ld[0], dev_Ld[1])];
	}
	if (threadIdx.y < RAD) {
		s_in[flatten(s_x, s_y - RAD, S_x, S_y)] = d_temp[flatten(k_x, k_y - RAD, dev_Ld[0], dev_Ld[1])];
		s_in[flatten(s_x, s_y + blockDim.y, S_x, S_y)] = d_temp[flatten(k_x, k_y + blockDim.y, dev_Ld[0], dev_Ld[1])];
	}
	
	// Calculate squared distance from pipe center
	float dSq = ((k_x - bc.x)*(k_x - bc.x) + (k_y - bc.y)*(k_y - bc.y));
	// If inside pipe, set temp to t_s and return
	if (dSq < bc.rad*bc.rad) {
		d_temp[k] = bc.t_s;
		return;
	}
	// If outside plate, set temp to t_a and return
	if ((k_x == 0 ) || (k_x == dev_Ld[0] - 1) || (k_y == 0 ) ||
		(k_x + k_y < bc.chamfer) || (k_x - k_y > dev_Ld[0] - bc.chamfer)) {
			d_temp[k] = bc.t_a;
			return;
	}
	// If point is below ground, set temp to t_g and return
	if (k_y == dev_Ld[1] - 1) {
		d_temp[k] = bc.t_g;
		return;
	}
	__syncthreads();
	// For all the remaining points, find temperature and set colors.
	float temp = 0.25f*(s_in[flatten(s_x - 1, s_y, S_x, S_y)] + 
				 s_in[flatten(s_x + 1,s_y,S_x,S_y)] + 
				 s_in[flatten(s_x, s_y - 1,S_x, S_y)] + 
				 s_in[flatten(s_x, s_y + 1, S_x, S_y)]);
	d_temp[k] = temp;

}


__global__ void float_to_char( uchar4* dev_out, const float* outSrc) {
	const int k_x = threadIdx.x + blockDim.x * blockIdx.x;
	const int k_y = threadIdx.y + blockDim.y * blockIdx.y;
	
	const int k   = k_x + k_y * blockDim.x*gridDim.x ; 

	dev_out[k].x = 0;
	dev_out[k].z = 0;
	dev_out[k].y = 0;
	dev_out[k].w = 255;


	const unsigned char intensity = clip((int) outSrc[k] ) ;
	dev_out[k].x = intensity ;       // higher temp -> more red
	dev_out[k].z = 255 - intensity ; // lower temp -> more blue
	
}


void kernelLauncher(uchar4 *d_out, float *d_temp, int w, int h, BC bc, dim3 M_in) {
	const dim3 gridSize(blocksNeeded(w, M_in.x), blocksNeeded(h, M_in.y));
	const size_t smSz = (M_in.x + 2 * RAD)*(M_in.y + 2 * RAD)*sizeof(float);

	tempKernel<<<gridSize, M_in, smSz>>>(d_temp, w, h , bc);

	float_to_char<<<gridSize,M_in>>>(d_out, d_temp) ; 
}

void kernelLauncher2(uchar4 *d_out, float *d_temp, int w, int h, BC bc, dim3 M_in) {
	const dim3 gridSize(blocksNeeded(w, M_in.x), blocksNeeded(h, M_in.y));
	const size_t smSz = (M_in.x + 2 * RAD)*(M_in.y + 2 * RAD)*sizeof(float);

	tempKernel2<<<gridSize, M_in, smSz>>>(d_temp, bc);

	float_to_char<<<gridSize,M_in>>>(d_out, d_temp) ; 
}


void resetTemperature(float *d_temp, int w, int h, BC bc, dim3 M_in) {
	const dim3 gridSize( blocksNeeded(w, M_in.x), blocksNeeded( h, M_in.y));

	resetKernel<<<gridSize, M_in>>>(d_temp,w,h,bc);
}


	
	
	
			
		
		
