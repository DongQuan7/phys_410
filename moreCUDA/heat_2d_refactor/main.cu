#include "hip/hip_runtime.h"
/* main.cpp
 * 2-dim. Laplace Eq. (heat eq.) by finite difference with shared memory
 * Ernest Yeung  ernestyalumni@gmail.com
 * 20160625
 */
#include <functional>

#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include "./physlib/heat_2d.h"
#include "./physlib/dev_R2grid.h"
#include "./physlib/R2grid.h"

#include "./commonlib/errors.h"
#include "./commonlib/tex_anim2d.h"
#include "./commonlib/finitediff.h"


#define GL_GLEXT_PROTOTYPES // needed for identifier glGenBuffer, glBindBuffer, glBufferData, glDeleteBuffers

#include <GL/glut.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h> // or #include "cuda_gl_interop.h"
#define ITERS_PER_RENDER 50

const float Deltat[1] { 0.0001f };

// physics
const int WIDTH  { 640 } ;
const int HEIGHT { 640 } ;

dim3 dev_L2 { static_cast<unsigned int>(WIDTH), 
				static_cast<unsigned int>(HEIGHT) };

dev_Grid2d dev_grid2d( dev_L2 );		

// graphics + physics

const dim3 M_i { 32 , 32  };

const int iters_per_render { 50 };

GPUAnim2dTex bitmap( WIDTH, HEIGHT );
GPUAnim2dTex* testGPUAnim2dTex = &bitmap; 


void make_render( int w, int h, int iters_per_render_in, GPUAnim2dTex* texmap  ) {
	uchar4 *d_out = 0;
	hipGraphicsMapResources(1, &texmap->cuda_pixbufferObj_resource, 0);
	hipGraphicsResourceGetMappedPointer((void **)&d_out, NULL,
		texmap->cuda_pixbufferObj_resource);

	for (int i = 0; i < iters_per_render_in; ++i) {
		kernelLauncher2(d_out, dev_grid2d.dev_temperature, w, h, bc, M_i );
//		kernelLauncher(d_out, dev_grid2d.dev_temperature, w, h, bc, M_i );
//		kernelLauncher3(d_out, dev_grid2d.dev_temperature, w, h, bc, M_i );
//		kernelLauncher4(d_out, dev_grid2d.dev_temperature, w, h, bc, M_i );
	}

	hipGraphicsUnmapResources(1, &texmap->cuda_pixbufferObj_resource, 0);
	
	char title[128];
	sprintf(title, "Temperature Visualizer - Iterations=%4d, "
				   "T_s=%3.0f, T_a=%3.0f, T_g=%3.0f",
				   iterationCount, bc.t_s, bc.t_a, bc.t_g);
	glutSetWindowTitle(title);
}	



std::function<void()> render = std::bind( make_render, WIDTH, HEIGHT, iters_per_render, testGPUAnim2dTex);	

std::function<void()> draw_texture = std::bind( make_draw_texture, WIDTH, HEIGHT) ;

void display() {
	render();
	draw_texture();
	glutSwapBuffers();
}



int main(int argc, char** argv) {
	// physics
	constexpr std::array<int,2> LdS {WIDTH, HEIGHT };
	constexpr std::array<float,2> ldS {1.f, 1.f };
	
	HANDLE_ERROR(
		hipMemcpyToSymbol(HIP_SYMBOL( dev_Deltat), Deltat, sizeof(float)*1,0,hipMemcpyHostToDevice) );

	const float heat_params[2] { 
//								 0.0061035f , 
								 0.00061035f,
								 1.f } ; // \kappa 
										// heat capacity for constant volume, per volume 

	HANDLE_ERROR(
		hipMemcpyToSymbol(HIP_SYMBOL( dev_heat_params), heat_params, sizeof(float)*2,0,hipMemcpyHostToDevice) );
	
	const int Ld_to_const[3] { LdS[0], LdS[1] } ;
	
	HANDLE_ERROR(
		hipMemcpyToSymbol(HIP_SYMBOL( dev_Ld), Ld_to_const, sizeof(int)*2,0,hipMemcpyHostToDevice) );
	
	Grid2d grid2d( LdS, ldS);
	
	const float hds[2] { grid2d.hd[0], grid2d.hd[1] } ;
	
	// sanity check
	std::cout << " hds : .x : " << hds[0] << " .y : " << hds[1] << std::endl;
	
//	set1DerivativeParameters(hds);
	set2DerivativeParameters(hds);
//	set3DerivativeParameters(hds);
//	set4DerivativeParameters(hds);
	
	
	resetTemperature( dev_grid2d.dev_temperature, WIDTH, HEIGHT, bc, M_i);
	
	
	printInstructions();


	testGPUAnim2dTex->initGLUT(&argc, argv);

	glutKeyboardFunc(keyboard_func);
	glutMouseFunc(mouse_func);
	glutIdleFunc(idle);

	glutDisplayFunc(display);

	testGPUAnim2dTex->initPixelBuffer();

	glutMainLoop();

	HANDLE_ERROR(
		hipFree( dev_grid2d.dev_temperature ) );

	return 0;
} 

	
