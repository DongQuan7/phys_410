#include "hip/hip_runtime.h"
/**
 * @file   : simpleDivergence_b.cu
 * @brief  : Simple warp divergence, with my editting
 * @author : Ernest Yeung <ernestyalumni@gmail.com>
 * @date   : 20170910  
 * @ref    :  https://github.com/deeperlearning/professional-cuda-c-programming/blob/master/examples/chapter03/simpleDivergence.cu
 * 
 * If you find this code useful, feel free to donate directly and easily at this direct PayPal link: 
 * 
 * https://www.paypal.com/cgi-bin/webscr?cmd=_donations&business=ernestsaveschristmas%2bpaypal%40gmail%2ecom&lc=US&item_name=ernestyalumni&currency_code=USD&bn=PP%2dDonationsBF%3abtn_donateCC_LG%2egif%3aNonHosted 
 * 
 * which won't go through a 3rd. party such as indiegogo, kickstarter, patreon.  
 * Otherwise, I receive emails and messages on how all my (free) material on 
 * physics, math, and engineering have helped students with their studies, 
 * and I know what it's like to not have money as a student, but love physics 
 * (or math, sciences, etc.), so I am committed to keeping all my material 
 * open-source and free, whether or not 
 * sufficiently crowdfunded, under the open-source MIT license: 
 *  feel free to copy, edit, paste, make your own versions, share, use as you wish.  
 *  Just don't be an asshole and not give credit where credit is due.  
 * Peace out, never give up! -EY
 * 
 * */
/* 
 * COMPILATION TIP
 * nvcc --relocatable-device-code true simpleDivergence.cu -o simpleDivergence.exe
 * 
 * */
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call)                                                         \
{                                                                           \
    const hipError_t error = call;                                         \
    if (error != hipSuccess)                                               \
    {                                                                       \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                       \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error));  \
        exit(1);                                                            \
    }                                                                       \
}                                                                           \

void checkResult(float *hostRef, float *gpuRef, const int N) {
    double epsilon = 1.0E-8;
    bool match = 1;
    for (int i=0; i<N; i++) {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }
    
    if (match) printf("Arrays match.\n\n");
}

void initialData(float *ip, int size) {
    // generate different seed for random number
    time_t t;
    srand((unsigned) time(&t));
    
    for (int i=0; i<size; i++) {
        ip[i] = (float)( rand() & 0xFF )/10.0f;
    }
}

void sumArraysOnHost(float *A, float *B, float *C, const int N) {
    for (int idx=0; idx<N; idx++) 
        C[idx] = A[idx] + B[idx];
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C) {
    int i = threadIdx.x;
    C[i] = A[i] + B[i]; 
}

int main(int argc, char **argv) {
    printf("%s Starting ... \n", argv[0]);
    
    // set up device
    int dev = 0;
    hipSetDevice(dev);
    
    // set updata size of vectors
    int nElem = 32;
    printf("Vector size %d\n", nElem);
    
    // malloc host memory
    size_t nBytes = nElem * sizeof(float);
    
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A     = (float *)malloc(nBytes);
    h_B     = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef  = (float *)malloc(nBytes);
    
    // initialize data at host side
    initialData(h_A, nElem);
    initialData(h_B, nElem);
    
    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);
    
    // malloc device global memory
    float *d_A, *d_B, *d_C;
    hipMalloc((float **)&d_A, nBytes);
    hipMalloc((float **)&d_B, nBytes);
    hipMalloc((float **)&d_C, nBytes);
    
    // transfer data from host to device
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
    
    // invoke kernel at host side
    dim3 block (nElem);
    dim3 grid (nElem/block.x);
    
    sumArraysOnGPU<<< grid, block >>>(d_A, d_B, d_C);
    printf("Execution configuration <<<%d, %d>>>\n", grid.x,block.x);
    
    // copy kernel result back to host side
    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);
    
    // add vector at host side for result checks
    sumArraysOnHost(h_A, h_B, hostRef, nElem);
    
    // check device results
    checkResult(hostRef, gpuRef, nElem);
    
    // free device global memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);
    
    return(0);
}
    
    