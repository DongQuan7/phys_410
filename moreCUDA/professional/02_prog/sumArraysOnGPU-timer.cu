#include "hip/hip_runtime.h"
/**
 * @file   : sumArraysOnGPU-timer.cu
 * @brief  : Measuring the vector summation kernel 
 * @author : Ernest Yeung <ernestyalumni@gmail.com>
 * @date   : 20170914
 * @ref    : John Cheng, Max Grossman, Ty McKercher. Professional CUDA C Programming. 1st Ed. Wrox. 2014
 * 		   : Ch. 6 Streams and Concurrency; pp. 271 
 * 
 * If you find this code useful, feel free to donate directly and easily at this direct PayPal link: 
 * 
 * https://www.paypal.com/cgi-bin/webscr?cmd=_donations&business=ernestsaveschristmas%2bpaypal%40gmail%2ecom&lc=US&item_name=ernestyalumni&currency_code=USD&bn=PP%2dDonationsBF%3abtn_donateCC_LG%2egif%3aNonHosted 
 * 
 * which won't go through a 3rd. party such as indiegogo, kickstarter, patreon.  
 * Otherwise, I receive emails and messages on how all my (free) material on 
 * physics, math, and engineering have helped students with their studies, 
 * and I know what it's like to not have money as a student, but love physics 
 * (or math, sciences, etc.), so I am committed to keeping all my material 
 * open-source and free, whether or not 
 * sufficiently crowdfunded, under the open-source MIT license: 
 * 	feel free to copy, edit, paste, make your own versions, share, use as you wish.  
 *  Just don't be an asshole and not give credit where credit is due.  
 * Peace out, never give up! -EY
 * 
 * */
/* 
 * COMPILATION TIP
 * g++ -std=c++14 FileIObin.cpp -o FileIObin.exe
 * 
 * */
#include <stdio.h>		// printf
#include <sys/time.h>	// gettimeofday

double cpuSecond() {
	struct timeval tp;
	gettimeofday(&tp,NULL);
	return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

void checkResult(float *hostRef, float *gpuRef, const int N) {
	double epsilon = 1.0E-8;
	bool match = 1;
	for (int i=0; i<N; i++) {
		if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
			match = 0;
			printf("Arrays do not match!\n");
			printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
			break;
		}
	}
	
	if (match) printf("Arrays match.\n\n");
}

void initialData(float *ip, int size) {
	// generate different seed for random number
	time_t t;
	srand((unsigned) time(&t)); 
	
	for (int i=0; i<size; i++) {
		ip[i] = (float)( rand() & 0xFF )/10.0f;
	}
}

void sumArraysOnHost(float *A, float *B, float *C, const int N) {
	for (int idx=0; idx<N; idx++) {
		C[idx] = A[idx] + B[idx];
	}
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N) {
	int i = threadIdx.x + blockDim.x *blockIdx.x;

	if (i >= N) {
		return; 
	}
	else {
		C[i] = A[i] + B[i];
	}
}


int main(int argc, char ** argv) {
	printf("%s Starting...\n", argv[0]);
	
	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);
	printf("Using Device %d: %s\n", dev, deviceProp.name);
	hipSetDevice(dev);
	
	// set up date size of vectors
	int nElem = 1<<24; 
	printf("Vector size %d\n", nElem);
	
	// malloc host memory
	size_t nBytes = nElem * sizeof(float);
	
	float *h_A, *h_B, *hostRef, *gpuRef;
	h_A		= (float *)malloc(nBytes);
	h_B 	= (float *)malloc(nBytes);
	hostRef	= (float *)malloc(nBytes);
	gpuRef	= (float *)malloc(nBytes);
	
	double iStart, iElaps;
	
	// initialize data at host side
	iStart = cpuSecond();
	initialData(h_A, nElem);
	initialData(h_B, nElem);
	iElaps = cpuSecond() - iStart;
	printf("initialData Time elapsed %f sec\n", iElaps);
	memset(hostRef, 0, nBytes);
	memset(gpuRef, 0, nBytes);
	
	// add vector at host side for result checks
	iStart = cpuSecond();
	sumArraysOnHost(h_A, h_B, hostRef, nElem);
	iElaps = cpuSecond() - iStart;
	printf("sumArraysOnHost Time elapsed %f sec\n", iElaps);

	
	// malloc device global memory
	float *d_A, *d_B, *d_C;
	hipMalloc((float**)&d_A, nBytes);
	hipMalloc((float**)&d_B, nBytes);
	hipMalloc((float**)&d_C, nBytes);
	
	// transfer data from host to device
	hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_C, gpuRef,nBytes,hipMemcpyHostToDevice);
	
	// invoke kernel at host side
	int iLen = 512;
	dim3 block (iLen);
	dim3 grid  ((nElem+block.x-1)/block.x);
	
	iStart = cpuSecond();
	sumArraysOnGPU<<<grid, block>>>(d_A, d_B, d_C, nElem);
	
	hipDeviceSynchronize();
	iElaps = cpuSecond() - iStart;
	printf("sumARraysOnGPU <<<%d,%d>>> Time elapsed %f" \
			"sec\n", grid.x, block.x, iElaps);
			
	// check kernel error
	hipGetLastError();		
			
	// copy kernel result back to host side
	hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);
	
	// check device results
	checkResult(hostRef, gpuRef, nElem);
	
	// free device global memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	
	// free host memory
	free(h_A);
	free(h_B);
	free(hostRef);
	free(gpuRef);
	
	hipDeviceReset();
	
	return(0);
}
	
