/**
 * @file   : commonmultiplestreams.cu
 * @brief  : Common pattern for dispatching CUDA operations to multiple streams 
 * @author : Ernest Yeung <ernestyalumni@gmail.com>
 * @date   : 20170904  
 * @ref    : http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#asynchronous-concurrent-execution
 * 		   : 3.2.5. Asynchronous Concurrent Execution of CUDA Toolkit v8.0, 3. Programming Interface  
 * 		   : John Cheng, Max Grossman, Ty McKercher. Professional CUDA C Programming. 1st Ed. Wrox. 2014
 * 		   : Ch. 6 Streams and Concurrency; pp. 271   
 * If you find this code useful, feel free to donate directly and easily at this direct PayPal link: 
 * 
 * https://www.paypal.com/cgi-bin/webscr?cmd=_donations&business=ernestsaveschristmas%2bpaypal%40gmail%2ecom&lc=US&item_name=ernestyalumni&currency_code=USD&bn=PP%2dDonationsBF%3abtn_donateCC_LG%2egif%3aNonHosted 
 * 
 * which won't go through a 3rd. party such as indiegogo, kickstarter, patreon.  
 * Otherwise, I receive emails and messages on how all my (free) material on 
 * physics, math, and engineering have helped students with their studies, 
 * and I know what it's like to not have money as a student, but love physics 
 * (or math, sciences, etc.), so I am committed to keeping all my material 
 * open-source and free, whether or not 
 * sufficiently crowdfunded, under the open-source MIT license: 
 * 	feel free to copy, edit, paste, make your own versions, share, use as you wish.  
 *  Just don't be an asshole and not give credit where credit is due.  
 * Peace out, never give up! -EY
 * 
 * */
/* 
 * COMPILATION TIP
 * nvcc -std=c++11 smart_ptrs_arith.cu -o smart_ptrs_arith.exe
 * 
 * */

#include <hip/hip_runtime.h>
#include <iostream>

int main(int argc, char *argv[]) {


}
