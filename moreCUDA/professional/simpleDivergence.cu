/**
 * @file   : simpleDivergence.cu
 * @brief  : Simple warp divergence
 * @author : Ernest Yeung <ernestyalumni@gmail.com>
 * @date   : 20170910  
 * @ref    :  https://github.com/deeperlearning/professional-cuda-c-programming/blob/master/examples/chapter03/simpleDivergence.cu
 * 
 * If you find this code useful, feel free to donate directly and easily at this direct PayPal link: 
 * 
 * https://www.paypal.com/cgi-bin/webscr?cmd=_donations&business=ernestsaveschristmas%2bpaypal%40gmail%2ecom&lc=US&item_name=ernestyalumni&currency_code=USD&bn=PP%2dDonationsBF%3abtn_donateCC_LG%2egif%3aNonHosted 
 * 
 * which won't go through a 3rd. party such as indiegogo, kickstarter, patreon.  
 * Otherwise, I receive emails and messages on how all my (free) material on 
 * physics, math, and engineering have helped students with their studies, 
 * and I know what it's like to not have money as a student, but love physics 
 * (or math, sciences, etc.), so I am committed to keeping all my material 
 * open-source and free, whether or not 
 * sufficiently crowdfunded, under the open-source MIT license: 
 * 	feel free to copy, edit, paste, make your own versions, share, use as you wish.  
 *  Just don't be an asshole and not give credit where credit is due.  
 * Peace out, never give up! -EY
 * 
 * */
/* 
 * COMPILATION TIP
 * nvcc --relocatable-device-code true simpleDivergence.cu -o simpleDivergence.exe
 * 
 * */
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * simpleDivergence demonstrates divergent code on the GPU and its impact on
 * performance and CUDA metrics.
 */

__global__ void mathKernel1(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    if (tid % 2 == 0)
    {
        ia = 100.0f;
    }
    else
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

__global__ void mathKernel2(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    if ((tid / warpSize) % 2 == 0)
    {
        ia = 100.0f;
    }
    else
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

__global__ void mathKernel3(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    bool ipred = (tid % 2 == 0);

    if (ipred)
    {
        ia = 100.0f;
    }

    if (!ipred)
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

__global__ void mathKernel4(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    int itid = tid >> 5;

    if (itid & 0x01 == 0)
    {
        ia = 100.0f;
    }
    else
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

__global__ void warmingup(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    if ((tid / warpSize) % 2 == 0)
    {
        ia = 100.0f;
    }
    else
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}


int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("%s using Device %d: %s\n", argv[0], dev, deviceProp.name);

    // set up data size
    int size = 64;
    int blocksize = 64;

    if(argc > 1) blocksize = atoi(argv[1]);

    if(argc > 2) size      = atoi(argv[2]);

    printf("Data size %d ", size);

    // set up execution configuration
    dim3 block (blocksize, 1);
    dim3 grid  ((size + block.x - 1) / block.x, 1);
    printf("Execution Configure (block %d grid %d)\n", block.x, grid.x);

    // allocate gpu memory
    float *d_C;
    size_t nBytes = size * sizeof(float);
    hipMalloc((float**)&d_C, nBytes);

    // create cuda event handles
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

	float gpu_time = 0.0f;

    // run a warmup kernel to remove overhead
    hipDeviceSynchronize();
	hipEventRecord(start, 0);
    warmingup<<<grid, block>>>(d_C);
    hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("warmup      <<< %4d %4d >>> elapsed %.10f msec \n", grid.x, block.x,
           gpu_time );
	hipGetLastError();

    // run kernel 1
	hipEventRecord(start, 0);
    mathKernel1<<<grid, block>>>(d_C);
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("mathKernel1 <<< %4d %4d >>> elapsed %.10f msec \n", grid.x, block.x,
           gpu_time );
	hipGetLastError();

    // run kernel 3
	hipEventRecord(start, 0);
    mathKernel2<<<grid, block>>>(d_C);
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("mathKernel2 <<< %4d %4d >>> elapsed %.10f msec \n", grid.x, block.x,
           gpu_time );
	hipGetLastError();

    // run kernel 3
	hipEventRecord(start, 0);
    mathKernel3<<<grid, block>>>(d_C);
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("mathKernel3 <<< %4d %4d >>> elapsed %.10f msec \n", grid.x, block.x,
           gpu_time );
    hipGetLastError();

    // run kernel 4
	hipEventRecord(start, 0);
    mathKernel4<<<grid, block>>>(d_C);
    hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("mathKernel4 <<< %4d %4d >>> elapsed %.10f msec \n", grid.x, block.x,
           gpu_time );
	hipGetLastError();

    // free gpu memory and reset divece
    hipFree(d_C);
    hipDeviceReset();
    return EXIT_SUCCESS;
}

  
