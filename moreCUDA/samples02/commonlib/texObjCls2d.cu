/* texObjCls2d.cu
 * texture memory Object class, 2-dim. 
 * Ernest Yeung  ernestyalumni@gmail.com
 * 20161115
 *  
 * compilation tip: (compile separately)
 * nvcc -std=c++11 -c ./commonlib/texObjCls2d.cu -o texObjCls2d.o
 * 
 */
#include "texObjCls2d.h"

// constructor
__host__ TexObj2d::TexObj2d( hipArray* cuArray ) : 
    texObj(0)
{
    // Specify texture
    memset(&(this->resDesc),0, sizeof(this->resDesc));
    (this->resDesc).resType = hipResourceTypeArray;
    (this->resDesc).res.array.array = cuArray;

    // Specify texture object parameters
    memset(&(this->texDesc), 0, sizeof(this->texDesc) );

    // set these 2 lines manually, either hipAddressModeClamp or hipAddressModeWrap
    (this->texDesc).addressMode[0] = hipAddressModeClamp ;
    (this->texDesc).addressMode[1] = hipAddressModeClamp ;
    // set filterMode manually; either hipFilterModePoint or hipFilterModeLinear
    (this->texDesc).filterMode     = hipFilterModePoint ;
    
    (this->texDesc).readMode       = hipReadModeElementType;
    // END specify texture object parameters

    // Create texture object
    checkCudaErrors(
        hipCreateTextureObject(&(this->texObj), &(this->resDesc), &(this->texDesc), NULL) );
    
}

// constructor
// Note: you're going to have to set the hipArray this texture object is associated to at a later time.
__host__ TexObj2d::TexObj2d(  ) : 
    texObj(0)
{
    // Specify texture
    memset(&(this->resDesc),0, sizeof(this->resDesc));
    (this->resDesc).resType = hipResourceTypeArray;

    // Specify texture object parameters
    memset(&(this->texDesc), 0, sizeof(this->texDesc) );

    // set these 2 lines manually, either hipAddressModeClamp or hipAddressModeWrap
    (this->texDesc).addressMode[0] = hipAddressModeClamp ;
    (this->texDesc).addressMode[1] = hipAddressModeClamp ;
    // set filterMode manually; either hipFilterModePoint or hipFilterModeLinear
    (this->texDesc).filterMode     = hipFilterModePoint ;
    
    (this->texDesc).readMode       = hipReadModeElementType;
    // END specify texture object parameters

    // Create texture object
    checkCudaErrors( 
        hipCreateTextureObject(&(this->texObj), &(this->resDesc), &(this->texDesc), NULL) );
    
}

// destructor
__host__ TexObj2d::~TexObj2d() {
    checkCudaErrors(
        hipDestroyTextureObject(this->texObj) ) ;
    
}