#include "hip/hip_runtime.h"
/*
 * tex1dlinearmem.cu
 * 
 * cf. http://www.math.ntu.edu.tw/~wwang/mtxcomp2010/download/cuda_04_ykhung.pdf
 * from Yukai Hung a0934147@gmail.com Math Dept. National Taiwan Univ.
 * 
 * Compilation
 * 
 * nvcc -arch=sm_20 interpolation_so.cu // non-normalized coordinates
 * nvcc -arch=sm_20 interpolation_so.cu -DNORMALIZED // normalized coordinates, incorrect answer
 * 
 * */

// declare texture reference (at file level)
texture<float,1,hipReadModeElementType> texreference;

__global__ void kernel(float* doarray, int size)
{
	int index;
	
	// calculate each thread global index
	index=blockIdx.x*blockDim.x+threadIdx.x;
	
	// fetch global memory through texture reference
	doarray[index]=tex1Dfetch(texreference,index);
	
	return;
}

int main(int argc, char** argv)
{
	int size=3200;
	
	float* harray;
	float* diarray;
	float* doarray;
	
	// allocate host and device memory
	harray=(float*)malloc(sizeof(float)*size);
	hipMalloc((void**)&diarray,sizeof(float)*size);
	hipMalloc((void**)&doarray,sizeof(float)*size);
	
	// initialize host array before usage
	for(int loop=0; loop<size;loop++)
		harray[loop]=(float)rand()/(float) (RAND_MAX-1);
		
	// copy array from host to device memory
	hipMemcpy(diarray,harray,sizeof(float)*size,hipMemcpyHostToDevice);
	
	// bind texture reference with linear memory
	hipBindTexture(0,texreference,diarray,sizeof(float)*size);
	
	// execute device kernel
	kernel<<<(int)ceil((float)size/64),64>>>(doarray,size);
	
	// unbind texture reference to free resource
	hipUnbindTexture(texreference);
	
	// copy result array from device to host memory
	hipMemcpy(harray,doarray,sizeof(float)*size,hipMemcpyDeviceToHost);
	
	// free host and device memory
	free(harray);
	hipFree(diarray);
	hipFree(doarray);
	
	return 0;
}
