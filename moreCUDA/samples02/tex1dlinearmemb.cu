#include "hip/hip_runtime.h"
/*
 * tex1dlinearmemb.cu
 * 
 * cf. http://www.math.ntu.edu.tw/~wwang/mtxcomp2010/download/cuda_04_ykhung.pdf
 * from Yukai Hung a0934147@gmail.com Math Dept. National Taiwan Univ.
 * 
 * Compilation
 * 
 * nvcc -std=c++11 tex1dlinearmemb.cu -o tex1dlinearmemb
 * 
 * */
#include <iostream> // std::cout
#include <limits>  // std::numeric_limits
// Note below: the "copy" immediate below each line of "original" code is for playing around
// manually change values to test stuff out!

// declare texture reference (at file level)
texture<float,1,hipReadModeElementType> texreference;
texture<float,hipTextureType1D,hipReadModeElementType> tex;

__global__ void kernel(float* doarray, int size)
{
	int index;
	
	// calculate each thread global index
	index=blockIdx.x*blockDim.x+threadIdx.x;
	
	// fetch global memory through texture reference
	doarray[index]=tex1Dfetch(texreference,index);
	
	return;
}
__global__ void kernel2(float* doarray, int size)
{
	// calculate each thread global index
	const int k_x = blockIdx.x*blockDim.x+threadIdx.x ;
	
	// fetch global memory through texture reference
	doarray[k_x]=tex1Dfetch(tex,k_x);
	
	return;
}



int main(int argc, char** argv)
{
	const int ARRAY_SIZE=3200;
	const int ARRAY_SIZE2=32; 

	constexpr const int M_in { 64 };
	
	float* harray;
	float* diarray;
	float* doarray;
	
	float xarray[ARRAY_SIZE2];
	float* dev_in;
	float* dev_out;
	
	// allocate host and device memory
	harray=(float*)malloc(sizeof(float)*ARRAY_SIZE);
	hipMalloc((void**)&diarray,sizeof(float)*ARRAY_SIZE);
	hipMalloc((void**)&doarray,sizeof(float)*ARRAY_SIZE);

	hipMalloc((void**)&dev_in, sizeof(float)*ARRAY_SIZE2);
	hipMalloc((void**)&dev_out, sizeof(float)*ARRAY_SIZE2);
	
	// initialize host array before usage
	for(int loop=0; loop<ARRAY_SIZE;loop++)
		harray[loop]=(float)rand()/(float) (RAND_MAX-1);

	for(int loop=0; loop<ARRAY_SIZE2;loop++) {
		xarray[loop]= (std::numeric_limits<float>::max()/(1000.f)) / pow(2,loop) ;
	}

	// sanity check: print out initial values:
	const int DISPLAY_SIZE = 22; // how may numbers you want to display, read out, or print out on screen
	static_assert( ARRAY_SIZE >= DISPLAY_SIZE, "ARRAY_SIZE needs to be equal or bigger than DISPLAY_SIZE");
	static_assert( ARRAY_SIZE2 >= DISPLAY_SIZE, "ARRAY_SIZE2 needs to be equal or bigger than DISPLAY_SIZE");

	std::cout << "Initially, " << std::endl;
	for (int i = 0; i < DISPLAY_SIZE; ++i) {
		std::cout << " " << harray[i] ; }
	std::cout << std::endl;

	std::cout << "\n and with xarray, initially, " << std::endl;
	for (int i = 0; i < DISPLAY_SIZE; ++i) {
		std::cout << " " << xarray[i] ; }
	std::cout << std::endl;


	std::cout << "numerical limit of float, minimum : " << std::numeric_limits<float>::min() << std::endl;
	std::cout << "numerical limit of float, maximum : " << std::numeric_limits<float>::max() << std::endl;
	std::cout << "numerical limit of float, lowest  : " << std::numeric_limits<float>::lowest() << std::endl;


		
	// copy array from host to device memory
	hipMemcpy(diarray,harray,sizeof(float)*ARRAY_SIZE,hipMemcpyHostToDevice);
	hipMemcpy(dev_in,xarray,sizeof(float)*ARRAY_SIZE2,hipMemcpyHostToDevice);

	// bind texture reference with linear memory
	hipBindTexture(0,texreference,diarray,sizeof(float)*ARRAY_SIZE);
	hipBindTexture(0,tex,dev_in,sizeof(float)*ARRAY_SIZE2);

	// execute device kernel
	kernel<<<(int)ceil((float)ARRAY_SIZE/64),64>>>(doarray,ARRAY_SIZE);
	kernel2<<<1,M_in>>>(dev_out,ARRAY_SIZE2);
	
	
	// unbind texture reference to free resource
	hipUnbindTexture(texreference);
	hipUnbindTexture(tex);
	
	// copy result array from device to host memory
	hipMemcpy(harray,doarray,sizeof(float)*ARRAY_SIZE,hipMemcpyDeviceToHost);
	hipMemcpy(xarray,dev_out,sizeof(float)*ARRAY_SIZE2,hipMemcpyDeviceToHost);

	// sanity check: print out, read out results
	std::cout << "After kernel, which has a tex1Dfetch, and hipMemcpy, " << std::endl;
	for (int i = 0; i < DISPLAY_SIZE; ++i) {
		std::cout << " " << harray[i] ; }
	std::cout << std::endl;

	std::cout << "\n And for xarray : " << std::endl;
	for (int i = 0; i < DISPLAY_SIZE; ++i) {
		std::cout << " " << xarray[i] ; }
	std::cout << std::endl;

	
	// free host and device memory
	free(harray);
	hipFree(diarray);
	hipFree(doarray);

	hipFree(dev_in);
	hipFree(dev_out);
	
	return 0;
}
