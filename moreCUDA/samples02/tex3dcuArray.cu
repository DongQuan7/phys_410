#include "hip/hip_runtime.h"
/*
 * tex3dcuArray.cu
 * 3-dimension cuda array example
 * 
 * cf. http://www.math.ntu.edu.tw/~wwang/mtxcomp2010/download/cuda_04_ykhung.pdf
 * from Yukai Hung a0934147@gmail.com Math Dept. National Taiwan Univ.
 * 
 * Compilation
 * 
 * nvcc -std=c++11 tex1dlinearmemb.cu -o tex1dlinearmemb
 * 
 * */
// declare texture reference (at file level)
texture<float,3,hipReadModeElementType> texreference;

__global__ void kernel(float* dmatrix, int size)
{
	int loop;
	int xindex;
	int yindex;
	int zindex;
	
	// calculate each thread global index
	xindex=blockIdx.x*blockDim.x+threadIdx.x;
	yindex=blockIdx.y*blockDim.y+threadIdx.y;
	
	for (loop=0;loop<size;loop++)
	{
		zindex=loop;
	
		// fetch cuda array through texture reference
		dmatrix[zindex*size*size + yindex*size+xindex]=
			tex3D(texreference,xindex,yindex,zindex);
	}
	return;
}

int main(int argc, char** argv)
{
	int size=256;

	dim3 blocknum;
	dim3 blocksize;
	
	float* hmatrix;
	float* dmatrix;
	
	hipArray* cudaarray;
	hipExtent volumesize;
	hipChannelFormatDesc channel;
	
	hipMemcpy3DParms copyparms={0};
	
	// allocate host and device memory
	hmatrix=(float*)malloc(sizeof(float)*size*size*size);
	hipMalloc((void**)&dmatrix,sizeof(float)*size*size*size);
	
	// initialize host array before usage
	for(int loop=0; loop<size*size*size;loop++)
		hmatrix[loop]=(float)rand()/(float) (RAND_MAX-1);

	// set cuda array volume size
	volumesize=make_hipExtent(size,size,size);	
		
	// create channel to describe data type
	channel=hipCreateChannelDesc<float>();	

	// allocate device memory for cuda array
	hipMalloc3DArray(&cudaarray,&channel,volumesize);

	// set cuda array copy parameters
	copyparms.extent=volumesize;
	copyparms.dstArray=cudaarray;
	copyparms.kind=hipMemcpyHostToDevice;
	
	copyparms.srcPtr= make_hipPitchedPtr((void*)hmatrix,sizeof(float)*size,size,size);
	
	hipMemcpy3D(&copyparms);

	// set texture filter mode property
	// use hipFilterModePoint or hipFilterModeLinear
	texreference.filterMode=hipFilterModePoint;

	// set texture address mode property
	// use hipAddressModeClamp or hipAddressModeWrap
	texreference.addressMode[0]=hipAddressModeWrap;
	texreference.addressMode[1]=hipAddressModeWrap;
	texreference.addressMode[2]=hipAddressModeClamp;

	// bind texture reference with cuda array
	hipBindTextureToArray(texreference,cudaarray,channel);

	
	blocksize.x=8;
	blocksize.y=8;
	blocksize.z=8;
	
	blocknum.x=(int)ceil((float)size/8);
	blocknum.y=(int)ceil((float)size/8);
	blocknum.z=(int)ceil((float)size/8);
	
		
	// execute device kernel
	kernel<<<blocknum,blocksize>>>(dmatrix,size);
	
	// unbind texture reference to free resource
	hipUnbindTexture(texreference);
	
	// copy result array from device to host memory
	const int bytes = sizeof(float)*size*size*size;
	hipMemcpy(hmatrix,dmatrix,bytes,hipMemcpyDeviceToHost);
	
	// free host and device memory
	free(hmatrix);
	hipFree(dmatrix);
	hipFreeArray(cudaarray);
	
	return 0;
}

