#include "hip/hip_runtime.h"
/* dev_R2grid.cu
 * R3 under discretization (discretize functor) to a grid
 * Ernest Yeung  ernestyalumni@gmail.com
 * 2016115
 * 
 * compilation tip: (compile separately)
 * nvcc -std=c++11 -c ./physlib/dev_R2grid.cu -o dev_R2grid.o
 * 
 */
#include "dev_R2grid.h"

//__constant__ int dev_Ld[2];

// constructor
__host__ dev_Grid2d::dev_Grid2d( dim3 Ld_in) : Ld(Ld_in)
{
	checkCudaErrors(
		hipMalloc((void**)&this->dev_f_in, this->NFLAT()*sizeof(float) ) );

	checkCudaErrors(
		hipMalloc((void**)&this->dev_f_out, this->NFLAT()*sizeof(float) ) );

	checkCudaErrors(
		hipMalloc((void**)&this->dev_f_constsrc, this->NFLAT()*sizeof(float) ) );


	checkCudaErrors(
		hipMalloc((void**)&this->dev_rho, this->NFLAT()*sizeof(float) ) );

	checkCudaErrors(
		hipMalloc((void**)&this->dev_rho_out, this->NFLAT()*sizeof(float) ) );

	checkCudaErrors(
		hipMalloc((void**)&this->dev_u, this->NFLAT()*sizeof(float2) ) );

	checkCudaErrors(
		hipMalloc((void**)&this->dev_u_out, this->NFLAT()*sizeof(float2) ) );

	checkCudaErrors(
		hipMalloc((void**)&this->dev_p, this->NFLAT()*sizeof(float2) ) );

	checkCudaErrors(
		hipMalloc((void**)&this->dev_p_out, this->NFLAT()*sizeof(float2) ) );

	checkCudaErrors(
		hipMalloc((void**)&this->dev_E, this->NFLAT()*sizeof(float) ) );

	checkCudaErrors(
		hipMalloc((void**)&this->dev_E_out, this->NFLAT()*sizeof(float) ) );

	
}

// destructor
__host__ dev_Grid2d::~dev_Grid2d() {

	checkCudaErrors(
		hipFree( this->dev_f_in ) );

	checkCudaErrors(
		hipFree( this->dev_f_out ) );

	checkCudaErrors(
		hipFree( this->dev_f_constsrc ) );


	checkCudaErrors(
		hipFree( this->dev_rho ) );

	checkCudaErrors(
		hipFree( this->dev_rho_out ) );

	checkCudaErrors(
		hipFree( this->dev_u ) );

	checkCudaErrors(
		hipFree( this->dev_u_out ) );

	checkCudaErrors(
		hipFree( this->dev_p ) );

	checkCudaErrors(
		hipFree( this->dev_p_out ) );

	checkCudaErrors(
		hipFree( this->dev_E ) );

	checkCudaErrors(
		hipFree( this->dev_E_out ) );


}


__host__ int dev_Grid2d :: NFLAT() {
	return Ld.x*Ld.y;
}	



